#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "hip/hip_runtime.h"

extern "C" {

int cuda_set_device(int i_gpu)
{
    hipError_t err = hipSetDevice(i_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaSetDevice: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_get_device_count(int *n_gpu)
{
    hipError_t err = hipGetDeviceCount(n_gpu);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaGetDeviceCount: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_device_synchronize()
{
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        printf("\n Error in cudaDeviceSynchronize: %s \n", hipGetErrorString(err));
    }

    return 0;
}

int cuda_malloc(intptr_t *a, size_t size)
{
    hipError_t err = hipMalloc((void **) a, size);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMalloc: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_free(intptr_t *a)
{
    hipError_t err = hipFree(a);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaFree: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

int cuda_memcpy(intptr_t *dest, intptr_t *src, size_t count, int dir)
{
    hipMemcpyKind dir2;

    switch (dir)
    {
        case 0:
            dir2 = hipMemcpyHostToDevice;
            break;
        case 1:
            dir2 = hipMemcpyDeviceToHost;
            break;
        case 2:
            dir2 = hipMemcpyDeviceToDevice;
            break;
    }

    hipError_t err = hipMemcpy(dest, src, count, dir2);

    if (err != hipSuccess)
    {
        printf("\n Error in cudaMemcpy: %s \n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

}
