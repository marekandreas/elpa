#include "hip/hip_runtime.h"
//    Copyright 2023, A. Marek
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//    This file was written by A. Marek, MPCDF

#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <stdint.h>
#include "config-f90.h"

//#include "../../../GPU/common_device_functions.h"

#define MAX_THREADS_PER_BLOCK 1024

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

// PETERDEBUG: clean up after merge. Uncommnent //#include "../../../GPU/common_device_functions.h" above
// construct a generic double/float/double_complex/float_complex from a double
template <typename T> __forceinline__ __device__ T elpaDeviceNumber(double number);
template <>  __forceinline__ __device__ double elpaDeviceNumber<double>(double number) {return number;}
template <>  __forceinline__ __device__ float  elpaDeviceNumber<float> (double number) {return (float) number;}
template <>  __forceinline__ __device__ hipDoubleComplex elpaDeviceNumber<hipDoubleComplex>(double number) {return make_hipDoubleComplex (number, 0.0);}
template <>  __forceinline__ __device__ hipFloatComplex elpaDeviceNumber<hipFloatComplex> (double number) {return make_hipFloatComplex ((float) number, 0.0);}


__global__ void cuda_copy_double_tmp2_c_kernel(double *tmp2_dev, double *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  //base 1 index
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];

}

extern "C" void cuda_copy_double_tmp2_c_FromC(double *tmp2_dev, double *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_double_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_tmp2_c_kernel(float *tmp2_dev, float *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_float_tmp2_c_FromC(float *tmp2_dev, float *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_float_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_double_complex_tmp2_c_kernel(hipDoubleComplex *tmp2_dev, hipDoubleComplex *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_double_complex_tmp2_c_FromC(double _Complex *tmp2_dev, double _Complex *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  hipDoubleComplex* tmp2_casted = (hipDoubleComplex*) tmp2_dev;
  hipDoubleComplex* c_casted = (hipDoubleComplex*) c_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_double_complex_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_complex_tmp2_c_kernel(hipFloatComplex *tmp2_dev, hipFloatComplex *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_float_complex_tmp2_c_FromC(float _Complex *tmp2_dev, float _Complex *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  hipFloatComplex* tmp2_casted = (hipFloatComplex*) tmp2_dev;
  hipFloatComplex* c_casted = (hipFloatComplex*) c_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_float_complex_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}



__global__ void cuda_copy_double_a_aux_bc_kernel(double *a_dev, double *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_double_a_aux_bc_FromC(double *a_dev, double *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_double_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}



__global__ void cuda_copy_float_a_aux_bc_kernel(float *a_dev, float *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_float_a_aux_bc_FromC(float *a_dev, float *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_float_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_double_complex_a_aux_bc_kernel(hipDoubleComplex *a_dev, hipDoubleComplex *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_double_complex_a_aux_bc_FromC(double _Complex *a_dev, double _Complex *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  
  hipDoubleComplex* a_dev_casted = (hipDoubleComplex*) a_dev;
  hipDoubleComplex* aux_bc_dev_casted = (hipDoubleComplex*) aux_bc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_double_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_float_complex_a_aux_bc_kernel(hipFloatComplex *a_dev, hipFloatComplex *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_float_complex_a_aux_bc_FromC(float _Complex *a_dev, float _Complex *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  
  hipFloatComplex* a_dev_casted = (hipFloatComplex*) a_dev;
  hipFloatComplex* aux_bc_dev_casted = (hipFloatComplex*) aux_bc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_float_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_double_aux_bc_aux_mat_kernel(double *aux_bc_dev, double *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk_mult, const int nblk) {
		
  //dim3 threadsPerBlock = dim3(1,1,1);
  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 blocks = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..lre-lrs+1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];

  //aux_mat(lrs:lre,nstor) = aux_bc(n_aux_bc+1:n_aux_bc+nvals)

}


extern "C" void cuda_copy_double_aux_bc_aux_mat_FromC(double *aux_bc_dev, double *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in , hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  //dim3 blocks = dim3(1,1,1);
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  //printf("C= lrs=%d, lre=%d, nstor=%d, n_aux_bc=%d, nvals=%d, l_rows=%d, nblk=%d, nblk_mult=%d \n", lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
  //printf("nvals=%d lre-lrs+1=%d \n", nvals, lre-lrs+1);

  //printf("Threads per Block %d\n",lre-lrs+1);
#ifdef WITH_GPU_STREAMS
  cuda_copy_double_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_double_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_aux_bc_aux_mat_kernel(float *aux_bc_dev, float *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_float_aux_bc_aux_mat_FromC(float *aux_bc_dev, float *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_float_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_double_complex_aux_bc_aux_mat_kernel(hipDoubleComplex *aux_bc_dev, hipDoubleComplex *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_double_complex_aux_bc_aux_mat_FromC(double _Complex *aux_bc_dev, double _Complex *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  hipDoubleComplex* aux_bc_dev_casted = (hipDoubleComplex*) aux_bc_dev;
  hipDoubleComplex* aux_mat_dev_casted = (hipDoubleComplex*) aux_mat_dev;


#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_double_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_float_complex_aux_bc_aux_mat_kernel(hipFloatComplex *aux_bc_dev, hipFloatComplex *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_float_complex_aux_bc_aux_mat_FromC(float _Complex *aux_bc_dev, float _Complex *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  hipFloatComplex* aux_bc_dev_casted = (hipFloatComplex*) aux_bc_dev;
  hipFloatComplex* aux_mat_dev_casted = (hipFloatComplex*) aux_mat_dev;


#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_float_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_aux_full_kernel(T *lhs_dev, T *rhs_dev, int l_rows, int l_cols, int lld_lhs, int lld_rhs) {

  // aux_a_full(1:l_rows,1:l_cols) = a(1:l_rows,1:l_cols)

  int i_loc = threadIdx.x; // 0..l_rows-1
  int j_loc = blockIdx.x ; // 0..l_cowl-1

  for (; j_loc < l_cols; j_loc += gridDim.x) {
    for (; i_loc < l_rows; i_loc += blockDim.x) {
      lhs_dev[i_loc+j_loc*lld_lhs] = rhs_dev[i_loc+j_loc*lld_rhs];
    }
  }
}

template <typename T>
void cuda_copy_aux_full(T *lhs_dev, T *rhs_dev, int *l_rows_in, int *l_cols_in, int *lld_lhs_in, int *lld_rhs_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int lld_lhs = *lld_lhs_in;
  int lld_rhs = *lld_rhs_in;
  int debug = *debug_in;

  dim3 blocks = dim3(l_cols,1,1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_aux_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(lhs_dev, rhs_dev, l_rows, l_cols, lld_lhs, lld_rhs);
#else
  cuda_copy_aux_full_kernel<<<blocks,threadsPerBlock>>>            (lhs_dev, rhs_dev, l_rows, l_cols, lld_lhs, lld_rhs);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_aux_full_FromC(char dataType, intptr_t lhs_dev, intptr_t rhs_dev,
                                         int *l_rows_in, int *l_cols_in, int *lld_lhs_in, int *lld_rhs_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_aux_full<double>((double *) lhs_dev, (double *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_aux_full<float> ((float  *) lhs_dev, (float  *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_aux_full<hipDoubleComplex>((hipDoubleComplex *) lhs_dev, (hipDoubleComplex *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_aux_full<hipFloatComplex> ((hipFloatComplex  *) lhs_dev, (hipFloatComplex  *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_full_kernel(T *a_dev, T *aux_mat_full_dev, int l_rows, int l_cols, int nblk_mult) {

  // aux_a_full(1:l_rows,1:l_cols) = a(1:l_rows,1:l_cols)
  // if (l_rows<nblk_mult) aux_a_full(l_rows+1:nblk_mult,1:l_cols) = 0
  // if (l_cols<nblk_mult) aux_a_full(1:l_rows,l_cols+1:nblk_mult) = 0
  // if (l_rows<nblk_mult .and. l_cols<nblk_mult) aux_a_full(l_rows+1:nblk_mult,l_cols+1:nblk_mult) = 0

  int i_loc = threadIdx.x; // 0..nblk_mult-1
  int j_loc = blockIdx.x ; // 0..nblk_mult-1

  T Zero = elpaDeviceNumber<T>(0.0);

  for (; j_loc < nblk_mult; j_loc += gridDim.x) {
    for (; i_loc < nblk_mult; i_loc += blockDim.x) {
      if (i_loc < l_rows && j_loc < l_cols) aux_mat_full_dev[i_loc+j_loc*nblk_mult] = a_dev[i_loc+j_loc*l_rows];
      else aux_mat_full_dev[i_loc+j_loc*nblk_mult] = Zero;
    }
  }
}

template <typename T>
void cuda_copy_and_set_zeros_aux_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult = *nblk_mult_in;
  int debug = *debug_in;

  dim3 blocks = dim3(nblk_mult,1,1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult);
#else
  cuda_copy_and_set_zeros_aux_full_kernel<<<blocks,threadsPerBlock>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_and_set_zeros_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_and_set_zeros_aux_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
}

//________________________________________________________________

// PETERDEBUG:  l_cols is unused, delete it
// also "variable "threadsPerBlock" was declared but never referenced" in this source file
template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_a_full_kernel(T *a_dev, T *aux_a_full_dev, int l_rows, int l_cols, int nblk_mult_cols, 
                                                          int nblk, int np_bc_fine, int np_cols_fine, int np_cols) {

  // do j_block_loc_fine = 0, nblk_mult_cols/nblk-1
  //   j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols
  //     aux_a_full(1:l_rows, 1+j_block_loc_fine*nblk: nblk+j_block_loc_fine*nblk) = &
  //              a(1:l_rows, 1+j_block_loc*nblk     : nblk+j_block_loc*nblk)
  //   enddo ! j_block_loc_fine
  //   if (mod(nblk_mult_cols,nblk) /= 0) then ! last incomplete nblk-block
  //     j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols
  //     aux_a_full(1:l_rows, 1+j_block_loc_fine*nblk: mod(nblk_mult_cols,nblk)+j_block_loc_fine*nblk) = &
  //              a(1:l_rows, 1+j_block_loc*nblk     : mod(nblk_mult_cols,nblk)+j_block_loc*nblk)
  //   endif
  // endif ! useGPU

  int i0 = threadIdx.x; // i  = 0..l_rows-1
  int dj0 = blockIdx.x; // dj = 0..nblk-1

  // Loop through full blocks
  int j_block_loc_fine = 0;
  for (; j_block_loc_fine < nblk_mult_cols/nblk; j_block_loc_fine++) 
    {
    int j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
    for (int dj = dj0; dj < nblk ; dj += gridDim.x)
      {
      for (int i = i0; i < l_rows; i += blockDim.x)
        {
        aux_a_full_dev[i + (dj+j_block_loc_fine*nblk)*l_rows] = a_dev[i + (dj+j_block_loc*nblk)*l_rows];
        }
      }
    }

  // Handle the last incomplete block if it exists
  if (nblk_mult_cols%nblk != 0) 
    {
    int j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
      for (int dj = dj0; dj < nblk_mult_cols%nblk ; dj += gridDim.x) 
        {
        for (int i = i0; i < l_rows; i += blockDim.x)
          {
          aux_a_full_dev[i + (dj+j_block_loc_fine*nblk)*l_rows] = a_dev[i + (dj+j_block_loc*nblk)*l_rows];
          }
        }
    }
}



template <typename T>
void cuda_copy_and_set_zeros_aux_a_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, 
                                        int *nblk_in, int *np_bc_fine_in, int *np_cols_fine_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult_cols = *nblk_mult_cols_in;
  int nblk = *nblk_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_cols = *np_cols_in;
  int debug = *debug_in;

  dim3 blocks = dim3(nblk, 1, 1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_a_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult_cols, 
                                                                                  nblk, np_bc_fine, np_cols_fine, np_cols);
#else
  cuda_copy_and_set_zeros_aux_a_full_kernel<<<blocks,threadsPerBlock>>>            (mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult_cols, 
                                                                                  nblk, np_bc_fine, np_cols_fine, np_cols);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_and_set_zeros_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_and_set_zeros_aux_a_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, int *nblk_in, int *np_bc_fine_in, int *np_cols_fine_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_a_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_a_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_a_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_a_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_b_full_kernel(T *b_dev, T *aux_b_full_dev, int l_rows, int l_cols, int nblk_mult, 
                                                          int nblk_mult_rows, int nblk, int np_fine, int np_rows_fine, int np_rows) {

  // do i_block_loc_fine = 0, nblk_mult_rows/nblk-1
  //   i_block_loc = (np_fine + i_block_loc_fine * np_rows_fine) / np_rows
  //   aux_b_full(1 + i_block_loc_fine * nblk : nblk + i_block_loc_fine * nblk, 1 : l_cols) = &
  //            b(1 + i_block_loc * nblk      : nblk + i_block_loc * nblk     , 1 : l_cols)
  // enddo ! i_block_loc_fine
  // if (mod(nblk_mult_rows, nblk) /= 0) then ! last incomplete nblk-block
  //   i_block_loc = (np_fine + i_block_loc_fine * np_rows_fine) / np_rows
  //   aux_b_full(1 + i_block_loc_fine * nblk : mod(nblk_mult_rows, nblk) + i_block_loc_fine * nblk, 1 : l_cols) = &
  //            b(1 + i_block_loc * nblk : mod(nblk_mult_rows, nblk) + i_block_loc * nblk, 1 : l_cols)
  // endif

  int di0 = threadIdx.x; // di = 0..nblk-1
  int j0  = blockIdx.x ; // j  = 0..l_cols-1

  // Loop through full blocks
  int i_block_loc_fine = 0;
  for (; i_block_loc_fine < nblk_mult_rows/nblk; i_block_loc_fine++) 
    {
    int i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    for (int j = j0; j < l_cols; j += gridDim.x)
      {
      for (int di = di0; di < nblk ; di += blockDim.x)
        {
        aux_b_full_dev[di + i_block_loc_fine*nblk + j*nblk_mult] = b_dev[di + i_block_loc*nblk + j*l_rows];
        }
      }
    }

  // Handle the last incomplete block if it exists
  if (nblk_mult_rows%nblk != 0)
    {
    int i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    for (int j = j0; j < l_cols; j += gridDim.x)
      {
      for (int di = di0; di < nblk_mult_rows%nblk ; di += blockDim.x)
        {
        aux_b_full_dev[di + i_block_loc_fine*nblk + j*nblk_mult] = b_dev[di + i_block_loc*nblk + j*l_rows];
        }
      }
    }
}

template <typename T>
void cuda_copy_and_set_zeros_aux_b_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_in, 
                                        int *nblk_mult_rows_in, int *nblk_in, int *np_fine_in, int *np_rows_fine_in, int *np_rows_in,
                                        int *SM_count_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult = *nblk_mult_in;
  int nblk_mult_rows = *nblk_mult_rows_in;
  int nblk = *nblk_in;
  int np_fine = *np_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_rows = *np_rows_in;
  int SM_count = *SM_count_in;
  int debug = *debug_in;

  dim3 blocks = dim3(SM_count, 1, 1); // PETERDEBUG what happens in the analogous Intel-GPU case? Here and in other analogous places
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

  // dim3 blocks = dim3(1,1,1);
  // dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_b_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult, 
                                                                                    nblk_mult_rows, nblk, np_fine, np_rows_fine, np_rows);
#else
  cuda_copy_and_set_zeros_aux_b_full_kernel<<<blocks,threadsPerBlock>>>            (mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult, 
                                                                                    nblk_mult_rows, nblk, np_fine, np_rows_fine, np_rows);
#endif
  
    if (debug)
      {
      hipDeviceSynchronize();
      hipError_t cuerr = hipGetLastError();
      if (cuerr != hipSuccess){
        printf("Error in executing cuda_copy_and_set_zeros_aux_b_full: %s\n",hipGetErrorString(cuerr));
      }
    }
  }

extern "C" void cuda_copy_and_set_zeros_aux_b_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_in, 
                                                       int *nblk_mult_rows_in, int *nblk_in, int *np_fine_in, int *np_rows_fine_in, int *np_rows_in,
                                                       int *SM_count_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_b_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_b_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_b_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_b_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_ccl_copy_buf_send_kernel(T *a_dev, T *buf_send_dev, int l_rows, int l_cols, int nblk_mult_rows, int lld_buf, int nblk,
                                              int m_blocks_loc_fine, int n_blocks_loc_fine, int np_fine, int np_bc_fine, 
                                              int np_rows_fine, int np_cols_fine, int np_rows, int np_cols) {
                                           
  // do j_block_loc_fine = 0, n_blocks_loc_fine_1 - 1
  //   j_block_loc = (np_bc_fine_1 + j_block_loc_fine*np_cols_fine)/np_cols
  //   nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk)

  //   m_blocks_loc_fine = (nblk_mult_rows+nblk-1)/nblk
  //   do i_block_loc_fine = 0, m_blocks_loc_fine - 1
  //     nblk_cut_row = min(nblk, nblk_mult_rows-i_block_loc_fine*nblk)
  //     i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows

  //     buf_send(1+i_block_loc_fine*nblk: nblk_cut_row+i_block_loc_fine*nblk,   &
  //               1+j_block_loc_fine*nblk: nblk_cut_col+j_block_loc_fine*nblk) = &
  //             a(1+i_block_loc     *nblk: nblk_cut_row+i_block_loc     *nblk,   &
  //               1+j_block_loc     *nblk: nblk_cut_col+j_block_loc     *nblk)
  //   enddo ! i_block_loc_fine
  // enddo ! j_block_loc_fine

  int i_block_loc, j_block_loc, nblk_cut_row, nblk_cut_col;

  int j_block_loc_fine = 0;
  for (; j_block_loc_fine < n_blocks_loc_fine; j_block_loc_fine++) 
    {
    j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
    nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk);

    m_blocks_loc_fine = (nblk_mult_rows+nblk-1)/nblk;
    int i_block_loc_fine = 0;
    for (; i_block_loc_fine < m_blocks_loc_fine; i_block_loc_fine++) 
      {
      nblk_cut_row = min(nblk, nblk_mult_rows-i_block_loc_fine*nblk);
      i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;

      for (int di = 0; di < nblk_cut_row; di++)
        {
        for (int dj = 0; dj < nblk_cut_col; dj++)
          {
          buf_send_dev[(di+i_block_loc_fine*nblk) + (dj+j_block_loc_fine*nblk)*lld_buf] 
               = a_dev[(di+i_block_loc*     nblk) + (dj+j_block_loc     *nblk)*l_rows];
          }
        }
      }
    }

}

template <typename T>
void cuda_ccl_copy_buf_send(T *a_dev, T *buf_send_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_rows_in, int *lld_buf_in, int *nblk_in,
                            int *m_blocks_loc_fine_in, int *n_blocks_loc_fine_in, int *np_fine_in, int *np_bc_fine_in, 
                            int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult_rows = *nblk_mult_rows_in;
  int lld_buf = *lld_buf_in;
  int nblk = *nblk_in;
  int m_blocks_loc_fine = *m_blocks_loc_fine_in;
  int n_blocks_loc_fine = *n_blocks_loc_fine_in;
  int np_fine = *np_fine_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_rows = *np_rows_in;
  int np_cols = *np_cols_in;
  int debug = *debug_in;

  dim3 blocks = dim3(1, 1, 1);
  dim3 threadsPerBlock = dim3(1, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_ccl_copy_buf_send_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_dev, buf_send_dev, l_rows, l_cols, nblk_mult_rows, lld_buf, nblk,
                                                                       m_blocks_loc_fine, n_blocks_loc_fine, np_fine, np_bc_fine, 
                                                                       np_rows_fine, np_cols_fine, np_rows, np_cols);
#else
  cuda_ccl_copy_buf_send_kernel<<<blocks,threadsPerBlock>>>(a_dev, buf_send_dev, l_rows, l_cols, nblk_mult_rows, lld_buf, nblk,
                                                           m_blocks_loc_fine, n_blocks_loc_fine, np_fine, np_bc_fine, 
                                                           np_rows_fine, np_cols_fine, np_rows, np_cols);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_ccl_copy_buf_send: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_ccl_copy_buf_send_FromC(char dataType, intptr_t a_dev, intptr_t buf_send_dev, 
                                             int *l_rows_in, int *l_cols_in, int *nblk_mult_rows_in, int *lld_buf_in, int *nblk_in,
                                             int *m_blocks_loc_fine_in, int *n_blocks_loc_fine_in, int *np_fine_in, int *np_bc_fine_in, 
                                             int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_ccl_copy_buf_send<double>((double *) a_dev, (double *) buf_send_dev, l_rows_in, l_cols_in, nblk_mult_rows_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='S') cuda_ccl_copy_buf_send<float> ((float  *) a_dev, (float  *) buf_send_dev, l_rows_in, l_cols_in, nblk_mult_rows_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='Z') cuda_ccl_copy_buf_send<hipDoubleComplex>((hipDoubleComplex *) a_dev, (hipDoubleComplex *) buf_send_dev, l_rows_in, l_cols_in, nblk_mult_rows_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='C') cuda_ccl_copy_buf_send<hipFloatComplex> ((hipFloatComplex  *) a_dev, (hipFloatComplex  *) buf_send_dev, l_rows_in, l_cols_in, nblk_mult_rows_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in,
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_ccl_copy_buf_recv_kernel(T *at_col_dev, T *buf_recv_dev, int l_rows, int l_cols, int nblk_mult_cols, int lld_buf, int nblk,
                                              int m_blocks_loc_fine, int n_blocks_loc_fine, int np_fine, int np_bc_fine, 
                                              int np_rows_fine, int np_cols_fine, int np_rows, int np_cols) {

  // do i_block_loc_fine = 0, m_blocks_loc_fine_1 - 1
  //   i_block_loc = (np_fine_1 + i_block_loc_fine*np_rows_fine)/np_rows

  //   nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk)

  //   do j_block_loc_fine = 0, n_blocks_loc_fine - 1
  //     nblk_cut_col = min(nblk, nblk_mult_cols-j_block_loc_fine*nblk)
  //     j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols
  //     at_col(1+i_block_loc     *nblk: nblk_cut_row+i_block_loc     *nblk,   &
  //             1+j_block_loc     *nblk: nblk_cut_col+j_block_loc     *nblk) = &
  // transpose(buf_recv(1+j_block_loc_fine*nblk: nblk_cut_col+j_block_loc_fine*nblk,   &
  //             1+i_block_loc_fine*nblk: nblk_cut_row+i_block_loc_fine*nblk))
  //   enddo ! j_block_loc_fine
  // enddo ! i_block_loc_fine
  printf("cuda_ccl_copy_buf_recv_kernel; m_blocks_loc_fine=%d\n", m_blocks_loc_fine); // PETERDEBUG

  int i_block_loc, j_block_loc, nblk_cut_row, nblk_cut_col;

  int i_block_loc_fine = 0;
  for (; i_block_loc_fine < m_blocks_loc_fine; i_block_loc_fine++) 
    {
    i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk);

    int j_block_loc_fine = 0;
    for (; j_block_loc_fine < n_blocks_loc_fine; j_block_loc_fine++) 
      {
      nblk_cut_col = min(nblk, nblk_mult_cols-j_block_loc_fine*nblk);
      j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;

      for (int di = 0; di < nblk_cut_row; di++)
        {
        for (int dj = 0; dj < nblk_cut_col; dj++)
          {
          // T temp = buf_recv_dev[(dj+j_block_loc_fine*nblk) + (di+i_block_loc_fine*nblk)*lld_buf];
          // printf("i_block_loc_fine = %d, j_block_loc_fine = %d, di = %d, dj = %d, temp=%f\n",
          //         i_block_loc_fine, j_block_loc_fine, di, dj, temp); // PETERDEBUG
          at_col_dev[(di+i_block_loc*     nblk) + (dj+j_block_loc*     nblk)*l_rows] 
      = buf_recv_dev[(dj+j_block_loc_fine*nblk) + (di+i_block_loc_fine*nblk)*lld_buf];
          }
        }
      }
    }

}

template <typename T>
void cuda_ccl_copy_buf_recv(T *at_col_dev, T *buf_recv_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, int *lld_buf_in, int *nblk_in,
                            int *m_blocks_loc_fine_in, int *n_blocks_loc_fine_in, int *np_fine_in, int *np_bc_fine_in, 
                            int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult_cols = *nblk_mult_cols_in;
  int lld_buf = *lld_buf_in;
  int nblk = *nblk_in;
  int m_blocks_loc_fine = *m_blocks_loc_fine_in;
  int n_blocks_loc_fine = *n_blocks_loc_fine_in;
  int np_fine = *np_fine_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_rows = *np_rows_in;
  int np_cols = *np_cols_in;
  int debug = *debug_in;

  dim3 blocks = dim3(1, 1, 1);
  dim3 threadsPerBlock = dim3(1, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_ccl_copy_buf_recv_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(at_col_dev, buf_recv_dev, l_rows, l_cols, nblk_mult_cols, lld_buf, nblk,
                                                                       m_blocks_loc_fine, n_blocks_loc_fine, np_fine, np_bc_fine, 
                                                                       np_rows_fine, np_cols_fine, np_rows, np_cols);
#else
  cuda_ccl_copy_buf_recv_kernel<<<blocks,threadsPerBlock>>>(at_col_dev, buf_recv_dev, l_rows, l_cols, nblk_mult_cols, lld_buf, nblk,
                                                           m_blocks_loc_fine, n_blocks_loc_fine, np_fine, np_bc_fine, 
                                                           np_rows_fine, np_cols_fine, np_rows, np_cols);
#endif
  
    if (debug)
      {
      hipDeviceSynchronize();
      hipError_t cuerr = hipGetLastError();
      if (cuerr != hipSuccess){
        printf("Error in executing cuda_ccl_copy_buf_recv: %s\n",hipGetErrorString(cuerr));
      }
    }
  }

extern "C" void cuda_ccl_copy_buf_recv_FromC(char dataType, intptr_t at_col_dev, intptr_t buf_recv_dev, 
                                             int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, int *lld_buf_in, int *nblk_in,
                                             int *m_blocks_loc_fine_in, int *n_blocks_loc_fine_in, int *np_fine_in, int *np_bc_fine_in, 
                                             int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_ccl_copy_buf_recv<double>((double *) at_col_dev, (double *) buf_recv_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='S') cuda_ccl_copy_buf_recv<float> ((float  *) at_col_dev, (float  *) buf_recv_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='Z') cuda_ccl_copy_buf_recv<hipDoubleComplex>((hipDoubleComplex *) at_col_dev, (hipDoubleComplex *) buf_recv_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
  if (dataType=='C') cuda_ccl_copy_buf_recv<hipFloatComplex> ((hipFloatComplex  *) at_col_dev, (hipFloatComplex  *) buf_recv_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, lld_buf_in, nblk_in,
                                                    m_blocks_loc_fine_in, n_blocks_loc_fine_in, np_fine_in, np_bc_fine_in,
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, debug_in, my_stream);
}