#include "hip/hip_runtime.h"
//    Copyright 2024, A. Marek
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//    This file was written by A. Marek, MPCDF

#include "config-f90.h"

#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <stdint.h>
#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)


__global__ void cuda_fill_tmp_arrays_double_kernel(int *idx1, int *p_col, int *coltyp, int *nnzu_val, int *nnzl_val, double *d1u, double *d1, double *zu, double *z, double *d1l, double *zl, const int na, const int np, const int na1, const int np_rem) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=0 && i < na1) {
      int index = idx1[i] - 1;
      if (p_col[index] == np_rem) {
        if ((coltyp[index] == 1) || (coltyp[index] == 2)) {
	  int nnzu = nnzu_val[(i) + na1 * (np-1)] -1 ;
	  d1u[nnzu] = d1[i];
	   zu[nnzu] =  z[i];
	}
        if ((coltyp[index] == 3) || (coltyp[index] == 2)) {
	  int nnzl = nnzl_val[(i) + na1 * (np-1)]-1;
	  d1l[nnzl] = d1[i];
	  zl[nnzl] =  z[i];
	}
      }
    }

}

extern "C" void cuda_fill_tmp_arrays_double_FromC(int *idx1_dev, int *p_col_dev, int *coltyp_dev, int *nnzu_val_dev, int *nnzl_val_dev, double *d1u_dev, double *d1_dev, double *zu_dev, double *z_dev, double *d1l_dev, double *zl_dev, int *na_in, int *np_in, int *na1_in, int *np_rem_in, hipStream_t  my_stream){
  int na = *na_in;
  int np = *np_in;
  int na1 = *na1_in;
  int np_rem = *np_rem_in;

  dim3 threadsPerBlock(1024);
  dim3 blocks((na1 + threadsPerBlock.x - 1) / threadsPerBlock.x);

#ifdef WITH_GPU_STREAMS
  cuda_fill_tmp_arrays_double_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(idx1_dev, p_col_dev, coltyp_dev, nnzu_val_dev, nnzl_val_dev, d1u_dev, d1_dev, zu_dev, z_dev, d1l_dev, zl_dev, na, np, na1, np_rem);
#else
  cuda_fill_tmp_arrays_double_kernel<<<blocks, threadsPerBlock>>>              (idx1_dev, p_col_dev, coltyp_dev, nnzu_val_dev, nnzl_val_dev, d1u_dev, d1_dev, zu_dev, z_dev, d1l_dev, zl_dev, na, np, na1, np_rem);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_fill_tmp_arrays_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

#ifdef WANT_SINGLE_PRECISION_REAL
__global__ void cuda_fill_tmp_arrays_float_kernel(int *idx1, int *p_col, int *coltyp, int *nnzu_val, int *nnzl_val, float *d1u, float *d1, float *zu, float *z, float *d1l, float *zl, const int na, const int np, const int na1, const int np_rem) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>=0 && i < na1) {
      int index = idx1[i] - 1;
      if (p_col[index] == np_rem) {
        if ((coltyp[index] == 1) || (coltyp[index] == 2)) {
	  int nnzu = nnzu_val[(i) + na1 * (np-1)] -1 ;
	  d1u[nnzu] = d1[i];
	   zu[nnzu] =  z[i];
	}
        if ((coltyp[index] == 3) || (coltyp[index] == 2)) {
	  int nnzl = nnzl_val[(i) + na1 * (np-1)]-1;
	  d1l[nnzl] = d1[i];
	  zl[nnzl] =  z[i];
	}
      }
    }

}

extern "C" void cuda_fill_tmp_arrays_float_FromC(int *idx1_dev, int *p_col_dev, int *coltyp_dev, int *nnzu_val_dev, int *nnzl_val_dev, float *d1u_dev, float *d1_dev, float *zu_dev, float *z_dev, float *d1l_dev, float *zl_dev, int *na_in, int *np_in, int *na1_in, int *np_rem_in, hipStream_t  my_stream){
  int na = *na_in;
  int np = *np_in;
  int na1 = *na1_in;
  int np_rem = *np_rem_in;

  dim3 threadsPerBlock(1024);
  dim3 blocks((na1 + threadsPerBlock.x - 1) / threadsPerBlock.x);

#ifdef WITH_GPU_STREAMS
  cuda_fill_tmp_arrays_float_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(idx1_dev, p_col_dev, coltyp_dev, nnzu_val_dev, nnzl_val_dev, d1u_dev, d1_dev, zu_dev, z_dev, d1l_dev, zl_dev, na, np, na1, np_rem);
#else
  cuda_fill_tmp_arrays_float_kernel<<<blocks, threadsPerBlock>>>              (idx1_dev, p_col_dev, coltyp_dev, nnzu_val_dev, nnzl_val_dev, d1u_dev, d1_dev, zu_dev, z_dev, d1l_dev, zl_dev, na, np, na1, np_rem);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_fill_tmp_arrays_float_kernel: %s\n",hipGetErrorString(cuerr));
  }
}
#endif


__global__ void cuda_copy_qtmp1_slice_to_q_double_kernel(double *q, double *qtmp1, int *l_col_out, int *p_col_out, int *ndef_c, int *p_col, int *idx2, int *idx, const int l_rqs, const int l_rqe, const int l_rows, const int matrixRows, const int gemm_dim_k, const int my_pcol, const int na1, const int np_rem, const int na) {
    int slice = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i>=0 && i < na) {
      if (slice >=0 && slice < l_rows) {
        int j = idx[i];
        if (j> na1) {
          int index3 = idx2[j-na1-1];
          if (p_col[index3-1] == np_rem) {
            if (p_col_out[i] == my_pcol) {
              if (slice >= 0 && slice < l_rows) {
                int ndef = ndef_c[i];
                int index2 = slice + gemm_dim_k * (ndef-1);
                int l_col = l_col_out[i];
                int index = l_rqs -1 + slice + matrixRows * (l_col-1);
                q[index] = qtmp1[index2];
              }
            }
          }
        }
      }

    }

}

extern "C" void cuda_copy_qtmp1_slice_to_q_double_FromC(double *q_dev, double *qtmp1_dev, int *l_col_out_dev, int *p_col_out_dev, int *ndef_c_dev, int *p_col_dev, int *idx2_dev, int *idx_dev, int *l_rqs_in, int *l_rqe_in, int *l_rows_in, int *matrixRows_in, int *gemm_dim_k_in, int *my_pcol_in, int *na1_in, int *np_rem_in, int *na_in, hipStream_t  my_stream){
  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int l_rows = *l_rows_in;
  int matrixRows = *matrixRows_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int my_pcol = *my_pcol_in;
  int na1 = *na1_in;
  int np_rem = *np_rem_in;
  int na = *na_in;

  dim3 threadsPerBlock(32,32);
  dim3 blocks((l_rows + threadsPerBlock.x - 1) / threadsPerBlock.x,(na + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_qtmp1_slice_to_q_double_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp1_dev, l_col_out_dev, p_col_out_dev, ndef_c_dev, p_col_dev, idx2_dev, idx_dev, l_rqs, l_rqe, l_rows, matrixRows, gemm_dim_k, my_pcol, na1, np_rem, na);
#else
  cuda_copy_qtmp1_slice_to_q_double_kernel<<<blocks, threadsPerBlock>>>(q_dev, q_dev, qtmp1_dev, l_col_out_dev, p_col_out_dev, ndef_c_dev, p_col_dev, idx2_dev, idx_dev, l_rqs, l_rqe, l_rows, matrixRows, gemm_dim_k, my_pcol, na1, np_rem, na);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_qtmp1_slice_to_q_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}



#ifdef WANT_SINGLE_PRECISION_REAL
__global__ void cuda_copy_qtmp1_slice_to_q_float_kernel(float *q, float *qtmp1, int *l_col_out, int *p_col_out, int *ndef_c, int *p_col, int *idx2, int *idx, const int l_rqs, const int l_rqe, const int l_rows, const int matrixRows, const int gemm_dim_k, const int my_pcol, const int na1, const int np_rem, const int na) {
    int slice = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i>=0 && i < na) {
      if (slice >=0 && slice < l_rows) {
        int j = idx[i];
        if (j> na1) {
          int index3 = idx2[j-na1-1];
          if (p_col[index3-1] == np_rem) {
            if (p_col_out[i] == my_pcol) {
              if (slice >= 0 && slice < l_rows) {
                int ndef = ndef_c[i];
                int index2 = slice + gemm_dim_k * (ndef-1);
                int l_col = l_col_out[i];
                int index = l_rqs -1 + slice + matrixRows * (l_col-1);
                q[index] = qtmp1[index2];
              }
            }
          }
        }
      }

    }

}

extern "C" void cuda_copy_qtmp1_slice_to_q_float_FromC(float *q_dev, float *qtmp1_dev, int *l_col_out_dev, int *p_col_out_dev, int *ndef_c_dev, int *p_col_dev, int *idx2_dev, int *idx_dev, int *l_rqs_in, int *l_rqe_in, int *l_rows_in, int *matrixRows_in, int *gemm_dim_k_in, int *my_pcol_in, int *na1_in, int *np_rem_in, int *na_in, hipStream_t  my_stream){
  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int l_rows = *l_rows_in;
  int matrixRows = *matrixRows_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int my_pcol = *my_pcol_in;
  int na1 = *na1_in;
  int np_rem = *np_rem_in;
  int na = *na_in;

  dim3 threadsPerBlock(32,32);
  dim3 blocks((l_rows + threadsPerBlock.x - 1) / threadsPerBlock.x,(na + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_qtmp1_slice_to_q_float_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp1_dev, l_col_out_dev, p_col_out_dev, ndef_c_dev, p_col_dev, idx2_dev, idx_dev, l_rqs, l_rqe, l_rows, matrixRows, gemm_dim_k, my_pcol, na1, np_rem, na);
#else
  cuda_copy_qtmp1_slice_to_q_float_kernel<<<blocks, threadsPerBlock>>>(q_dev, q_dev, qtmp1_dev, l_col_out_dev, p_col_out_dev, ndef_c_dev, p_col_dev, idx2_dev, idx_dev, l_rqs, l_rqe, l_rows, matrixRows, gemm_dim_k, my_pcol, na1, np_rem, na);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_qtmp1_slice_to_q_float_kernel: %s\n",hipGetErrorString(cuerr));
  }
}
#endif

__global__ void cuda_copy_q_slice_to_qtmp2_double_kernel(double *q, double *qtmp2, int *idxq1, int *l_col_out, const int l_rows, const int l_rqs, const int l_rqe, const int matrixRows, const int matrixCols, const int gemm_dim_k, const int gemm_dim_m, const int ns, const int ncnt, const int indx, const int indx2, const int na) {
    int j = blockIdx.x * blockDim.x + threadIdx.x; // 1.._l_rows
    int ii = blockIdx.y * blockDim.y + threadIdx.y + 1; // 1.._l_rows



    if (ii >=1 && ii < ncnt+1) {
      if (j >= 0 && j < l_rows) {
        int idx_2= ii+1+ns-1;
        int idx = idxq1[idx_2-1] ; 
        int k = l_col_out[idx-1]; 

        int index2 = j + l_rqs-1 + matrixRows * (k-1);
        int index  = j + gemm_dim_k * (ii-1);
        qtmp2[index]=q[index2];
      } 
    }
}


extern "C" void cuda_copy_q_slice_to_qtmp2_double_FromC(double *q_dev, double *qtmp2_dev, int *idxq1, int *l_col_out, int *l_rows_in, int *l_rqs_in, int *l_rqe_in, int *matrixRows_in, int *matrixCols_in, int *gemm_dim_k_in, int *gemm_dim_m_in, int *ns_in, int * ncnt_in, int *indx_in, int *indx2_in, int *na_in, hipStream_t  my_stream){
  int l_rows = *l_rows_in;
  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int matrixRows = *matrixRows_in;
  int matrixCols = *matrixCols_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int gemm_dim_m = *gemm_dim_m_in;
  int ns = *ns_in;
  int ncnt = *ncnt_in;
  int indx = *indx_in;
  int indx2 = *indx2_in;
  int na = *na_in;

  dim3 threadsPerBlock(1024);
  dim3 blocks((l_rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (ncnt + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_q_slice_to_qtmp2_double_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp2_dev, idxq1, l_col_out, l_rows, l_rqs, l_rqe, matrixRows, matrixCols, gemm_dim_k, gemm_dim_m, ns, ncnt, indx, indx2, na);
#else
  cuda_copy_q_slice_to_qtmp2_double_kernel<<<blocks, threadsPerBlock>>>(q_dev, qtmp2_dev, idxq1, l_col_out, l_rows, l_rqs, l_rqe, matrixRows, matrixCols, gemm_dim_k, gemm_dim_m, ns, indx, indx2, na);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_q_slice_to_qtmp2_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


#ifdef WANT_SINGLE_PRECISION_REAL
__global__ void cuda_copy_q_slice_to_qtmp2_float_kernel(float *q, float *qtmp2, int *idxq1, int *l_col_out, const int l_rows, const int l_rqs, const int l_rqe, const int matrixRows, const int matrixCols, const int gemm_dim_k, const int gemm_dim_m, const int ns, const int ncnt, const int indx, const int indx2, const int na) {
    int j = blockIdx.x * blockDim.x + threadIdx.x; // 1.._l_rows
    int ii = blockIdx.y * blockDim.y + threadIdx.y + 1; // 1.._l_rows



    if (ii >=1 && ii < ncnt+1) {
      if (j >= 0 && j < l_rows) {
        int idx_2= ii+1+ns-1;
        int idx = idxq1[idx_2-1] ; 
        int k = l_col_out[idx-1]; 

        int index2 = j + l_rqs-1 + matrixRows * (k-1);
        int index  = j + gemm_dim_k * (ii-1);
        qtmp2[index]=q[index2];
      } 
    }
}


extern "C" void cuda_copy_q_slice_to_qtmp2_float_FromC(float *q_dev, float *qtmp2_dev, int *idxq1, int *l_col_out, int *l_rows_in, int *l_rqs_in, int *l_rqe_in, int *matrixRows_in, int *matrixCols_in, int *gemm_dim_k_in, int *gemm_dim_m_in, int *ns_in, int * ncnt_in, int *indx_in, int *indx2_in, int *na_in, hipStream_t  my_stream){
  int l_rows = *l_rows_in;
  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int matrixRows = *matrixRows_in;
  int matrixCols = *matrixCols_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int gemm_dim_m = *gemm_dim_m_in;
  int ns = *ns_in;
  int ncnt = *ncnt_in;
  int indx = *indx_in;
  int indx2 = *indx2_in;
  int na = *na_in;

  dim3 threadsPerBlock(1024);
  dim3 blocks((l_rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (ncnt + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_q_slice_to_qtmp2_float_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp2_dev, idxq1, l_col_out, l_rows, l_rqs, l_rqe, matrixRows, matrixCols, gemm_dim_k, gemm_dim_m, ns, ncnt, indx, indx2, na);
#else
  cuda_copy_q_slice_to_qtmp2_float_kernel<<<blocks, threadsPerBlock>>>(q_dev, qtmp2_dev, idxq1, l_col_out, l_rows, l_rqs, l_rqe, matrixRows, matrixCols, gemm_dim_k, gemm_dim_m, ns, indx, indx2, na);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_q_slice_to_qtmp2_float_kernel: %s\n",hipGetErrorString(cuerr));
  }
}
#endif

__global__ void cuda_fill_ev_double_kernel(double *ev, double *tmp, double *d1u, double *dbase, double *ddiff, double *zu, double *ev_scale, int *idxq1, int *idx, const int na, const int gemm_dim_l, const int gemm_dim_m, const int nnzu, const int ns, const int ncnt) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (k >=0 && k< nnzu && k < na) {
      if (i>=1 && i < ncnt+1) {
        if (nnzu >= 1) {
          int idx_2= i+1+ns-1;
          int indx = idxq1[idx_2-1] ;
          int j = idx[indx-1];

          tmp[k] = d1u[k] - dbase[j-1];
          tmp[k] = tmp[k] + ddiff[j-1];
          ev[k + gemm_dim_l*(i-1)] = zu[k] / tmp[k] * ev_scale[j-1];
      }
    }
}

}


extern "C" void cuda_fill_ev_double_FromC(double *ev_dev, double *tmp_dev, double *d1u_dev, double *dbase_dev, double *ddiff_dev, double *zu_dev, double *ev_scale_dev, int *idxq1_dev, int  *idx_dev, int *na_in, int *gemm_dim_l_in, int *gemm_dim_m_in, int *nnzu_in, int *ns_in, int *ncnt_in, hipStream_t  my_stream){
  int na = *na_in;
  int gemm_dim_l = *gemm_dim_l_in;
  int gemm_dim_m = *gemm_dim_m_in;
  int nnzu = *nnzu_in;
  int ns = *ns_in;
  int ncnt = *ncnt_in;


  dim3 threadsPerBlock(32,32);
  dim3 blocks((nnzu + threadsPerBlock.x - 1) / threadsPerBlock.x, (ncnt + threadsPerBlock.y - 1) / threadsPerBlock.y);

  if (nnzu >= 1) {
#ifdef WITH_GPU_STREAMS
  cuda_fill_ev_double_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(ev_dev, tmp_dev, d1u_dev, dbase_dev, ddiff_dev, zu_dev, ev_scale_dev, idxq1_dev, idx_dev, na, gemm_dim_l, gemm_dim_m, nnzu, ns, ncnt);
#else
  cuda_fill_ev_double_kernel<<<blocks, threadsPerBlock>>>(ev_dev, tmp_dev, d1u_dev, dbase_dev, ddiff_dev, zu_dev, ev_scale_dev, idxq1_dev, idx_dev, na, gemm_dim_l, gemm_dim_m, nnzu, ns, ncnt);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_fill_ev_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
  }
}



#ifdef WANT_SINGLE_PRECISION_REAL
__global__ void cuda_fill_ev_float_kernel(float *ev, float *tmp, float *d1u, float *dbase, float *ddiff, float *zu, float *ev_scale, int *idxq1, int *idx, const int na, const int gemm_dim_l, const int gemm_dim_m, const int nnzu, const int ns, const int ncnt) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (k >=0 && k< nnzu && k < na) {
      if (i>=1 && i < ncnt+1) {
        if (nnzu >= 1) {
          int idx_2= i+1+ns-1;
          int indx = idxq1[idx_2-1] ;
          int j = idx[indx-1];

          tmp[k] = d1u[k] - dbase[j-1];
          tmp[k] = tmp[k] + ddiff[j-1];
          ev[k + gemm_dim_l*(i-1)] = zu[k] / tmp[k] * ev_scale[j-1];
      }
    }
}

}


extern "C" void cuda_fill_ev_float_FromC(float *ev_dev, float *tmp_dev, float *d1u_dev, float *dbase_dev, float *ddiff_dev, float *zu_dev, float *ev_scale_dev, int *idxq1_dev, int  *idx_dev, int *na_in, int *gemm_dim_l_in, int *gemm_dim_m_in, int *nnzu_in, int *ns_in, int *ncnt_in, hipStream_t  my_stream){
  int na = *na_in;
  int gemm_dim_l = *gemm_dim_l_in;
  int gemm_dim_m = *gemm_dim_m_in;
  int nnzu = *nnzu_in;
  int ns = *ns_in;
  int ncnt = *ncnt_in;


  dim3 threadsPerBlock(32,32);
  dim3 blocks((nnzu + threadsPerBlock.x - 1) / threadsPerBlock.x, (ncnt + threadsPerBlock.y - 1) / threadsPerBlock.y);

  if (nnzu >= 1) {
#ifdef WITH_GPU_STREAMS
  cuda_fill_ev_float_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(ev_dev, tmp_dev, d1u_dev, dbase_dev, ddiff_dev, zu_dev, ev_scale_dev, idxq1_dev, idx_dev, na, gemm_dim_l, gemm_dim_m, nnzu, ns, ncnt);
#else
  cuda_fill_ev_float_kernel<<<blocks, threadsPerBlock>>>(ev_dev, tmp_dev, d1u_dev, dbase_dev, ddiff_dev, zu_dev, ev_scale_dev, idxq1_dev, idx_dev, na, gemm_dim_l, gemm_dim_m, nnzu, ns, ncnt);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_fill_ev_float_kernel: %s\n",hipGetErrorString(cuerr));
  }
  }
}
#endif

__global__ void cuda_copy_qtmp2_slice_to_q_double_kernel(double *q, double *qtmp2, int *idx1q, int *l_col_out, const int l_rqs, const int l_rqe, const int l_rows,  const int ncnt, const int gemm_dim_k, const int matrixRows, const int ns) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (j>=0 && j<l_rows) {
      if (i>=0 && i< ncnt) {
        int idx = idx1q[((i+1) + ns) - 1];
        int l_col = l_col_out[idx-1];

        q[j+l_rqs - 1 + matrixRows * (l_col-1)] = qtmp2[j+gemm_dim_k*i];
      }
    }

}

extern "C" void cuda_copy_qtmp2_slice_to_q_double_FromC(double *q_dev, double *qtmp2_dev, int *idx1q_dev, int *l_col_out_dev, int *l_rqs_in, int *l_rqe_in, int *l_rows_in, int *ncnt_in, int *gemm_dim_k_in, int *matrixRows_in, int *ns_in, hipStream_t  my_stream){

  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int l_rows = *l_rows_in;
  int ncnt = *ncnt_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int matrixRows = *matrixRows_in;
  int ns = *ns_in;

  dim3 threadsPerBlock(32,32);
  dim3 blocks((ncnt + threadsPerBlock.x - 1) / threadsPerBlock.x, (l_rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_qtmp2_slice_to_q_double_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp2_dev, idx1q_dev, l_col_out_dev, l_rqs, l_rqe, l_rows, ncnt, gemm_dim_k, matrixRows, ns);
#else
  cuda_copy_qtmp2_slice_to_q_double_kernel<<<blocks, threadsPerBlock>>>(q_dev, qtmp2_dev, idx1q_dev, l_col_out_dev, l_rqs, l_rqe, l_rows, ncnt, gemm_dim_k, matrixRows, ns);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_qtmp2_slice_to_q_double_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


#ifdef WANT_SINGLE_PRECISION_REAL
__global__ void cuda_copy_qtmp2_slice_to_q_float_kernel(float *q, float *qtmp2, int *idx1q, int *l_col_out, const int l_rqs, const int l_rqe, const int l_rows,  const int ncnt, const int gemm_dim_k, const int matrixRows, const int ns) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (j>=0 && j<l_rows) {
      if (i>=0 && i< ncnt) {
        int idx = idx1q[((i+1) + ns) - 1];
        int l_col = l_col_out[idx-1];

        q[j+l_rqs - 1 + matrixRows * (l_col-1)] = qtmp2[j+gemm_dim_k*i];
      }
    }

}

extern "C" void cuda_copy_qtmp2_slice_to_q_float_FromC(float *q_dev, float *qtmp2_dev, int *idx1q_dev, int *l_col_out_dev, int *l_rqs_in, int *l_rqe_in, int *l_rows_in, int *ncnt_in, int *gemm_dim_k_in, int *matrixRows_in, int *ns_in, hipStream_t  my_stream){

  int l_rqs = *l_rqs_in;
  int l_rqe = *l_rqe_in;
  int l_rows = *l_rows_in;
  int ncnt = *ncnt_in;
  int gemm_dim_k = *gemm_dim_k_in;
  int matrixRows = *matrixRows_in;
  int ns = *ns_in;

  dim3 threadsPerBlock(32,32);
  dim3 blocks((ncnt + threadsPerBlock.x - 1) / threadsPerBlock.x, (l_rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

#ifdef WITH_GPU_STREAMS
  cuda_copy_qtmp2_slice_to_q_float_kernel<<<blocks, threadsPerBlock, 0, my_stream>>>(q_dev, qtmp2_dev, idx1q_dev, l_col_out_dev, l_rqs, l_rqe, l_rows, ncnt, gemm_dim_k, matrixRows, ns);
#else
  cuda_copy_qtmp2_slice_to_q_float_kernel<<<blocks, threadsPerBlock>>>(q_dev, qtmp2_dev, idx1q_dev, l_col_out_dev, l_rqs, l_rqe, l_rows, ncnt, gemm_dim_k, matrixRows, ns);
#endif

  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing cuda_copy_qtmp2_slice_to_q_float_kernel: %s\n",hipGetErrorString(cuerr));
  }
}
#endif
