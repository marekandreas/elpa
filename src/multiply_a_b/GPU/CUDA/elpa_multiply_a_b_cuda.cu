#include "hip/hip_runtime.h"
//    Copyright 2023, A. Marek
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//    This file was written by A. Marek, MPCDF

// PETERDEBUG: split this file in two .h files and one .cu file
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <stdint.h>
#include "config-f90.h"
//#include "../../../GPU/common_device_functions.h"

#define MAX_THREADS_PER_BLOCK 1024

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

// PETERDEBUG: clean up after merge. Uncommnent //#include "../../../GPU/common_device_functions.h" above
// construct a generic double/float/double_complex/float_complex from a double
template <typename T> __forceinline__ __device__ T elpaDeviceNumber(double number);
template <>  __forceinline__ __device__ double elpaDeviceNumber<double>(double number) {return number;}
template <>  __forceinline__ __device__ float  elpaDeviceNumber<float> (double number) {return (float) number;}
template <>  __forceinline__ __device__ hipDoubleComplex elpaDeviceNumber<hipDoubleComplex>(double number) {return make_hipDoubleComplex (number, 0.0);}
template <>  __forceinline__ __device__ hipFloatComplex elpaDeviceNumber<hipFloatComplex> (double number) {return make_hipFloatComplex ((float) number, 0.0);}


__global__ void cuda_copy_double_tmp2_c_kernel(double *tmp2_dev, double *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  //base 1 index
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];

}

extern "C" void cuda_copy_double_tmp2_c_FromC(double *tmp2_dev, double *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_double_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_tmp2_c_kernel(float *tmp2_dev, float *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_float_tmp2_c_FromC(float *tmp2_dev, float *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_float_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_dev, c_dev, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_double_complex_tmp2_c_kernel(hipDoubleComplex *tmp2_dev, hipDoubleComplex *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_double_complex_tmp2_c_FromC(double _Complex *tmp2_dev, double _Complex *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  hipDoubleComplex* tmp2_casted = (hipDoubleComplex*) tmp2_dev;
  hipDoubleComplex* c_casted = (hipDoubleComplex*) c_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_double_complex_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_complex_tmp2_c_kernel(hipFloatComplex *tmp2_dev, hipFloatComplex *c_dev, const int nr_done, const int nstor, const int lcs, const int lce, const int ldc, const int ldcCols){

  //dim3 blocks = dim3(lce-lcs+1,1,1);
  //dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  int i_index    = threadIdx.x +1; // range 1..nstor
  int j_index = blockIdx.x + 1; // range 1..lce-lse+1
  //c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(lcs-1+j_index-1)];
  c_dev[nr_done+(i_index-1) + ldc*(lcs-1+j_index-1)] = tmp2_dev[0+(i_index-1)+nstor*(j_index-1)];
}

extern "C" void cuda_copy_float_complex_tmp2_c_FromC(float _Complex *tmp2_dev, float _Complex *c_dev, int *nr_done_in, int *nstor_in, int *lcs_in, int *lce_in, int *ldc_in, int *ldcCols_in, hipStream_t my_stream) { 
		
  int nr_done = *nr_done_in;   
  int nstor = *nstor_in;
  int lcs = *lcs_in;
  int lce = *lce_in;
  int ldc = *ldc_in;
  int ldcCols = *ldcCols_in;

  dim3 blocks = dim3(lce-lcs+1,1,1);
  dim3 threadsPerBlock = dim3(nr_done+nstor-(nr_done+1)+1,1,1);

  hipFloatComplex* tmp2_casted = (hipFloatComplex*) tmp2_dev;
  hipFloatComplex* c_casted = (hipFloatComplex*) c_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_tmp2_c_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#else
  cuda_copy_float_complex_tmp2_c_kernel<<<blocks,threadsPerBlock>>>(tmp2_casted, c_casted, nr_done, nstor, lcs, lce, ldc, ldcCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_tmp2_c_kernel: %s\n",hipGetErrorString(cuerr));
  }
}



__global__ void cuda_copy_double_a_aux_bc_kernel(double *a_dev, double *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_double_a_aux_bc_FromC(double *a_dev, double *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_double_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}



__global__ void cuda_copy_float_a_aux_bc_kernel(float *a_dev, float *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_float_a_aux_bc_FromC(float *a_dev, float *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_float_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev, aux_bc_dev, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_double_complex_a_aux_bc_kernel(hipDoubleComplex *a_dev, hipDoubleComplex *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_double_complex_a_aux_bc_FromC(double _Complex *a_dev, double _Complex *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  
  hipDoubleComplex* a_dev_casted = (hipDoubleComplex*) a_dev;
  hipDoubleComplex* aux_bc_dev_casted = (hipDoubleComplex*) aux_bc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_double_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_float_complex_a_aux_bc_kernel(hipFloatComplex *a_dev, hipFloatComplex *aux_bc_dev, const int n_aux_bc, const int nvals, const int lrs, const int lre, const int noff, const int nblk, const int n, const int l_rows, const int lda, const int ldaCols){

  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = blockIdx.x +1; // range 1..lre-lrs+1
  int j_index = threadIdx.x + 1; // range 1..1
  aux_bc_dev[(n_aux_bc+1-1)+(i_index-1)] = a_dev[(lrs-1)+(i_index-1)+lda*(noff*nblk+n-1)];
}

extern "C" void cuda_copy_float_complex_a_aux_bc_FromC(float _Complex *a_dev, float _Complex *aux_bc_dev, int *n_aux_bc_in, int *nvals_in, int *lrs_in, int *lre_in, int *noff_in, int *nblk_in, int *n_in, int *l_rows_in, int *lda_in, int *ldaCols_in, hipStream_t my_stream) { 
		
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int lrs = *lrs_in;
  int lre = *lre_in;
  int noff = *noff_in;
  int nblk = *nblk_in;
  int n = *n_in;
  int l_rows = *l_rows_in;
  int lda = *lda_in;
  int ldaCols = *ldaCols_in;

  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  
  hipFloatComplex* a_dev_casted = (hipFloatComplex*) a_dev;
  hipFloatComplex* aux_bc_dev_casted = (hipFloatComplex*) aux_bc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#else
  cuda_copy_float_complex_a_aux_bc_kernel<<<blocks,threadsPerBlock>>>(a_dev_casted, aux_bc_dev_casted, n_aux_bc, nvals, lrs, lre, noff, nblk, n, l_rows, lda, ldaCols);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_a_aux_bc_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_double_aux_bc_aux_mat_kernel(double *aux_bc_dev, double *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk_mult, const int nblk) {
		
  //dim3 threadsPerBlock = dim3(1,1,1);
  //dim3 blocks = dim3(lre-lrs+1,1,1);
  //dim3 blocks = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..lre-lrs+1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];

  //aux_mat(lrs:lre,nstor) = aux_bc(n_aux_bc+1:n_aux_bc+nvals)

}


extern "C" void cuda_copy_double_aux_bc_aux_mat_FromC(double *aux_bc_dev, double *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in , hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  //dim3 blocks = dim3(1,1,1);
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  //printf("C= lrs=%d, lre=%d, nstor=%d, n_aux_bc=%d, nvals=%d, l_rows=%d, nblk=%d, nblk_mult=%d \n", lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
  //printf("nvals=%d lre-lrs+1=%d \n", nvals, lre-lrs+1);

  //printf("Threads per Block %d\n",lre-lrs+1);
#ifdef WITH_GPU_STREAMS
  cuda_copy_double_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_double_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_float_aux_bc_aux_mat_kernel(float *aux_bc_dev, float *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_float_aux_bc_aux_mat_FromC(float *aux_bc_dev, float *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_float_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev, aux_mat_dev, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_double_complex_aux_bc_aux_mat_kernel(hipDoubleComplex *aux_bc_dev, hipDoubleComplex *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_double_complex_aux_bc_aux_mat_FromC(double _Complex *aux_bc_dev, double _Complex *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  hipDoubleComplex* aux_bc_dev_casted = (hipDoubleComplex*) aux_bc_dev;
  hipDoubleComplex* aux_mat_dev_casted = (hipDoubleComplex*) aux_mat_dev;


#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_double_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}


__global__ void cuda_copy_float_complex_aux_bc_aux_mat_kernel(hipFloatComplex *aux_bc_dev, hipFloatComplex *aux_mat_dev, const int lrs, const int lre, const int nstor, const int n_aux_bc, const int nvals, const int l_rows, const int nblk, const int nblk_mult) {
		
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  int i_index    = threadIdx.x +1; // range 1..lre-lrs+1
  int j_index = blockIdx.x + 1; // range 1..1
  aux_mat_dev[lrs-1+(j_index-1)+l_rows*(nstor-1)] = aux_bc_dev[n_aux_bc+(j_index-1)];
}

extern "C" void cuda_copy_float_complex_aux_bc_aux_mat_FromC(float _Complex *aux_bc_dev, float _Complex *aux_mat_dev, int *lrs_in, int *lre_in, int *nstor_in, int *n_aux_bc_in, int *nvals_in, int *l_rows_in, int *nblk_in, int *nblk_mult_in, hipStream_t my_stream) {
		


  int lrs = *lrs_in;
  int lre = *lre_in;
  int nstor = *nstor_in;
  int n_aux_bc = *n_aux_bc_in;   
  int nvals = *nvals_in;
  int l_rows = *l_rows_in;
  int nblk_mult = *nblk_mult_in;
  int nblk = *nblk_in;
  
  dim3 blocks = dim3(lre-lrs+1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

  hipFloatComplex* aux_bc_dev_casted = (hipFloatComplex*) aux_bc_dev;
  hipFloatComplex* aux_mat_dev_casted = (hipFloatComplex*) aux_mat_dev;


#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#else
  cuda_copy_float_complex_aux_bc_aux_mat_kernel<<<blocks,threadsPerBlock>>>(aux_bc_dev_casted, aux_mat_dev_casted, lrs, lre, nstor, n_aux_bc, nvals, l_rows, nblk, nblk_mult);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_aux_bc_aux_mat_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_aux_full_kernel(T *lhs_dev, T *rhs_dev, int l_rows, int l_cols, int lld_lhs, int lld_rhs) {

  // aux_a_full(1:l_rows,1:l_cols) = a(1:l_rows,1:l_cols)

  int i_loc = threadIdx.x; // 0..l_rows-1
  int j_loc = blockIdx.x ; // 0..l_cowl-1

  for (; j_loc < l_cols; j_loc += gridDim.x) {
    for (; i_loc < l_rows; i_loc += blockDim.x) {
      lhs_dev[i_loc+j_loc*lld_lhs] = rhs_dev[i_loc+j_loc*lld_rhs];
    }
  }
}

template <typename T>
void cuda_copy_aux_full(T *lhs_dev, T *rhs_dev, int *l_rows_in, int *l_cols_in, int *lld_lhs_in, int *lld_rhs_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int lld_lhs = *lld_lhs_in;
  int lld_rhs = *lld_rhs_in;
  int debug = *debug_in;

  dim3 blocks = dim3(l_cols,1,1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_aux_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(lhs_dev, rhs_dev, l_rows, l_cols, lld_lhs, lld_rhs);
#else
  cuda_copy_aux_full_kernel<<<blocks,threadsPerBlock>>>            (lhs_dev, rhs_dev, l_rows, l_cols, lld_lhs, lld_rhs);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_aux_full_FromC(char dataType, intptr_t lhs_dev, intptr_t rhs_dev,
                                         int *l_rows_in, int *l_cols_in, int *lld_lhs_in, int *lld_rhs_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_aux_full<double>((double *) lhs_dev, (double *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_aux_full<float> ((float  *) lhs_dev, (float  *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_aux_full<hipDoubleComplex>((hipDoubleComplex *) lhs_dev, (hipDoubleComplex *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_aux_full<hipFloatComplex> ((hipFloatComplex  *) lhs_dev, (hipFloatComplex  *) rhs_dev, l_rows_in, l_cols_in, lld_lhs_in, lld_rhs_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_full_kernel(T *a_dev, T *aux_mat_full_dev, int l_rows, int l_cols, int nblk_mult) {

  // aux_a_full(1:l_rows,1:l_cols) = a(1:l_rows,1:l_cols)
  // if (l_rows<nblk_mult) aux_a_full(l_rows+1:nblk_mult,1:l_cols) = 0
  // if (l_cols<nblk_mult) aux_a_full(1:l_rows,l_cols+1:nblk_mult) = 0
  // if (l_rows<nblk_mult .and. l_cols<nblk_mult) aux_a_full(l_rows+1:nblk_mult,l_cols+1:nblk_mult) = 0

  int i_loc = threadIdx.x; // 0..nblk_mult-1
  int j_loc = blockIdx.x ; // 0..nblk_mult-1

  T Zero = elpaDeviceNumber<T>(0.0);

  for (; j_loc < nblk_mult; j_loc += gridDim.x) {
    for (; i_loc < nblk_mult; i_loc += blockDim.x) {
      if (i_loc < l_rows && j_loc < l_cols) aux_mat_full_dev[i_loc+j_loc*nblk_mult] = a_dev[i_loc+j_loc*l_rows];
      else aux_mat_full_dev[i_loc+j_loc*nblk_mult] = Zero;
    }
  }
}

template <typename T>
void cuda_copy_and_set_zeros_aux_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult = *nblk_mult_in;
  int debug = *debug_in;

  dim3 blocks = dim3(nblk_mult,1,1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult);
#else
  cuda_copy_and_set_zeros_aux_full_kernel<<<blocks,threadsPerBlock>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_and_set_zeros_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_and_set_zeros_aux_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, debug_in, my_stream);
}

//________________________________________________________________

// PETERDEBUG:  l_cols is unused, delete it
// also "variable "threadsPerBlock" was declared but never referenced" in this source file
template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_a_full_kernel(T *a_dev, T *aux_a_full_dev, int l_rows, int l_cols, int nblk_mult_cols,
                                                          int nblk, int np_bc_fine, int np_cols_fine, int np_cols) {

  // do j_block_loc_fine = 0, nblk_mult_cols/nblk-1
  //   j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols
  //     aux_a_full(1:l_rows, 1+j_block_loc_fine*nblk: nblk+j_block_loc_fine*nblk) = &
  //              a(1:l_rows, 1+j_block_loc*nblk     : nblk+j_block_loc*nblk)
  //   enddo ! j_block_loc_fine
  //   if (mod(nblk_mult_cols,nblk) /= 0) then ! last incomplete nblk-block
  //     j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols
  //     aux_a_full(1:l_rows, 1+j_block_loc_fine*nblk: mod(nblk_mult_cols,nblk)+j_block_loc_fine*nblk) = &
  //              a(1:l_rows, 1+j_block_loc*nblk     : mod(nblk_mult_cols,nblk)+j_block_loc*nblk)
  //   endif
  // endif ! useGPU

  int i0 = threadIdx.x; // i  = 0..l_rows-1
  int dj0 = blockIdx.x; // dj = 0..nblk-1

  // Loop through full blocks
  int j_block_loc_fine = 0;
  for (; j_block_loc_fine < nblk_mult_cols/nblk; j_block_loc_fine++) 
    {
    int j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
    for (int dj = dj0; dj < nblk ; dj += gridDim.x)
      {
      for (int i = i0; i < l_rows; i += blockDim.x)
        {
        aux_a_full_dev[i + (dj+j_block_loc_fine*nblk)*l_rows] = a_dev[i + (dj+j_block_loc*nblk)*l_rows];
        }
      }
    }

  // Handle the last incomplete block if it exists
  if (nblk_mult_cols%nblk != 0) 
    {
    int j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
      for (int dj = dj0; dj < nblk_mult_cols%nblk ; dj += gridDim.x) 
        {
        for (int i = i0; i < l_rows; i += blockDim.x)
          {
          aux_a_full_dev[i + (dj+j_block_loc_fine*nblk)*l_rows] = a_dev[i + (dj+j_block_loc*nblk)*l_rows];
          }
        }
    }
}



template <typename T>
void cuda_copy_and_set_zeros_aux_a_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, 
                                        int *nblk_in, int *np_bc_fine_in, int *np_cols_fine_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult_cols = *nblk_mult_cols_in;
  int nblk = *nblk_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_cols = *np_cols_in;
  int debug = *debug_in;

  dim3 blocks = dim3(nblk, 1, 1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_a_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult_cols,
                                                                                    nblk, np_bc_fine, np_cols_fine, np_cols);
#else
  cuda_copy_and_set_zeros_aux_a_full_kernel<<<blocks,threadsPerBlock>>>            (mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult_cols,
                                                                                    nblk, np_bc_fine, np_cols_fine, np_cols);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_copy_and_set_zeros_aux_full: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_copy_and_set_zeros_aux_a_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_cols_in, int *nblk_in, int *np_bc_fine_in, int *np_cols_fine_in, int *np_cols_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_a_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_a_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_a_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_a_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_cols_in, nblk_in, np_bc_fine_in, np_cols_fine_in, np_cols_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_b_full_kernel(T *b_dev, T *aux_b_full_dev, int l_rows, int l_cols, int nblk_mult, 
                                                          int nblk_mult_rows, int nblk, int np_fine, int np_rows_fine, int np_rows) {

  // do i_block_loc_fine = 0, nblk_mult_rows/nblk-1
  //   i_block_loc = (np_fine + i_block_loc_fine * np_rows_fine) / np_rows
  //   aux_b_full(1 + i_block_loc_fine * nblk : nblk + i_block_loc_fine * nblk, 1 : l_cols) = &
  //            b(1 + i_block_loc * nblk      : nblk + i_block_loc * nblk     , 1 : l_cols)
  // enddo ! i_block_loc_fine
  // if (mod(nblk_mult_rows, nblk) /= 0) then ! last incomplete nblk-block
  //   i_block_loc = (np_fine + i_block_loc_fine * np_rows_fine) / np_rows
  //   aux_b_full(1 + i_block_loc_fine * nblk : mod(nblk_mult_rows, nblk) + i_block_loc_fine * nblk, 1 : l_cols) = &
  //            b(1 + i_block_loc * nblk : mod(nblk_mult_rows, nblk) + i_block_loc * nblk, 1 : l_cols)
  // endif

  int di0 = threadIdx.x; // di = 0..nblk-1
  int j0  = blockIdx.x ; // j  = 0..l_cols-1

  // Loop through full blocks
  int i_block_loc_fine = 0;
  for (; i_block_loc_fine < nblk_mult_rows/nblk; i_block_loc_fine++) 
    {
    int i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    for (int j = j0; j < l_cols; j += gridDim.x)
      {
      for (int di = di0; di < nblk ; di += blockDim.x)
        {
        aux_b_full_dev[di + i_block_loc_fine*nblk + j*nblk_mult] = b_dev[di + i_block_loc*nblk + j*l_rows];
        }
      }
    }

  // Handle the last incomplete block if it exists
  if (nblk_mult_rows%nblk != 0)
    {
    int i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    for (int j = j0; j < l_cols; j += gridDim.x)
      {
      for (int di = di0; di < nblk_mult_rows%nblk ; di += blockDim.x)
        {
        aux_b_full_dev[di + i_block_loc_fine*nblk + j*nblk_mult] = b_dev[di + i_block_loc*nblk + j*l_rows];
        }
      }
    }
}

template <typename T>
void cuda_copy_and_set_zeros_aux_b_full(T *mat_dev, T *aux_mat_full_dev, int *l_rows_in, int *l_cols_in, int *nblk_mult_in, 
                                        int *nblk_mult_rows_in, int *nblk_in, int *np_fine_in, int *np_rows_fine_in, int *np_rows_in,
                                        int *SM_count_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int nblk_mult = *nblk_mult_in;
  int nblk_mult_rows = *nblk_mult_rows_in;
  int nblk = *nblk_in;
  int np_fine = *np_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_rows = *np_rows_in;
  int SM_count = *SM_count_in;
  int debug = *debug_in;

  dim3 blocks = dim3(SM_count, 1, 1); // PETERDEBUG what happens in the analogous Intel-GPU case? Here and in other analogous places
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

  // dim3 blocks = dim3(1,1,1);
  // dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_and_set_zeros_aux_b_full_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult, 
                                                                                    nblk_mult_rows, nblk, np_fine, np_rows_fine, np_rows);
#else
  cuda_copy_and_set_zeros_aux_b_full_kernel<<<blocks,threadsPerBlock>>>            (mat_dev, aux_mat_full_dev, l_rows, l_cols, nblk_mult, 
                                                                                    nblk_mult_rows, nblk, np_fine, np_rows_fine, np_rows);
#endif
  
    if (debug)
      {
      hipDeviceSynchronize();
      hipError_t cuerr = hipGetLastError();
      if (cuerr != hipSuccess){
        printf("Error in executing cuda_copy_and_set_zeros_aux_b_full: %s\n",hipGetErrorString(cuerr));
      }
    }
  }

extern "C" void cuda_copy_and_set_zeros_aux_b_full_FromC(char dataType, intptr_t mat_dev, intptr_t aux_mat_full_dev,
                                                       int *l_rows_in, int *l_cols_in, int *nblk_mult_in, 
                                                       int *nblk_mult_rows_in, int *nblk_in, int *np_fine_in, int *np_rows_fine_in, int *np_rows_in,
                                                       int *SM_count_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_copy_and_set_zeros_aux_b_full<double>((double *) mat_dev, (double *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='S') cuda_copy_and_set_zeros_aux_b_full<float> ((float  *) mat_dev, (float  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='Z') cuda_copy_and_set_zeros_aux_b_full<hipDoubleComplex>((hipDoubleComplex *) mat_dev, (hipDoubleComplex *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
  if (dataType=='C') cuda_copy_and_set_zeros_aux_b_full<hipFloatComplex> ((hipFloatComplex  *) mat_dev, (hipFloatComplex  *) aux_mat_full_dev, l_rows_in, l_cols_in, nblk_mult_in, nblk_mult_rows_in, nblk_in, np_fine_in, np_rows_fine_in, np_rows_in, SM_count_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_ccl_copy_buf_send_kernel(T *a_dev, T *buf_send_dev, int l_rows, int l_cols, int lld_buf, int nblk,
                                              int i_block_loc_fine_max, int j_block_loc_fine_max, int np_fine, int np_bc_fine, 
                                              int np_rows_fine, int np_cols_fine, int np_rows, int np_cols) {
                                           
  // ! The nested loop is symmetric wrt to i,j, so we use the rigid order of indices for convenience of copying
  // do j_block_loc_fine = 0, j_block_loc_fine_max
  //   j_block_loc = (np_t + j_block_loc_fine*np_cols_fine)/np_cols
  //   nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk)

  //   do i_block_loc_fine = 0, i_block_loc_fine_max
  //     i_block_loc = (np + i_block_loc_fine*np_rows_fine)/np_rows
  //     nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk)

  //     buf_send(1+ i_block_loc_fine*nblk: nblk_cut_row + i_block_loc_fine*nblk,   &
  //               1+ j_block_loc_fine*nblk: nblk_cut_col + j_block_loc_fine*nblk) = &
  //             a(1+ i_block_loc     *nblk: nblk_cut_row + i_block_loc     *nblk,   &
  //               1+ j_block_loc     *nblk: nblk_cut_col + j_block_loc     *nblk)
  //   enddo ! i_block_loc_fine
  // enddo ! j_block_loc_fine

  int di0 = threadIdx.x; // di = 0..nblk_cut_row-1
  int dj0 = blockIdx.x ; // dj = 0..nblk_cut_col-1

  int i_block_loc, j_block_loc, nblk_cut_row, nblk_cut_col;

  int j_block_loc_fine = 0;
  for (; j_block_loc_fine <= j_block_loc_fine_max; j_block_loc_fine++) 
    {
    // printf("j_block_loc_fine = %d\n", j_block_loc_fine); // PETERDEBUG
    j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
    nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk);

    int i_block_loc_fine = 0;
    for (; i_block_loc_fine <= i_block_loc_fine_max; i_block_loc_fine++) 
      {
      i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
      nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk);
      
      for (int dj = dj0; dj < nblk_cut_col; dj += gridDim.x)
        {
        for (int di = di0; di < nblk_cut_row; di += blockDim.x)
          {
          buf_send_dev[(di+i_block_loc_fine*nblk) + (dj+j_block_loc_fine*nblk)*lld_buf] 
               = a_dev[(di+i_block_loc*     nblk) + (dj+j_block_loc     *nblk)*l_rows];
          }
        }
      }
    }
}

template <typename T>
void cuda_ccl_copy_buf_send(T *a_dev, T *buf_send_dev, int *l_rows_in, int *l_cols_in, int *lld_buf_in, int *nblk_in,
                            int *i_block_loc_fine_max_in, int *j_block_loc_fine_max_in, int *np_fine_in, int *np_bc_fine_in, 
                            int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *SM_count_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int lld_buf = *lld_buf_in;
  int nblk = *nblk_in;
  int i_block_loc_fine_max = *i_block_loc_fine_max_in;
  int j_block_loc_fine_max = *j_block_loc_fine_max_in;
  int np_fine = *np_fine_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_rows = *np_rows_in;
  int np_cols = *np_cols_in;
  int SM_count = *SM_count_in;
  int debug = *debug_in;

  dim3 blocks = dim3(SM_count, 1, 1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_ccl_copy_buf_send_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_dev, buf_send_dev, l_rows, l_cols, lld_buf, nblk,
                                                                        i_block_loc_fine_max, j_block_loc_fine_max, np_fine, np_bc_fine, 
                                                                        np_rows_fine, np_cols_fine, np_rows, np_cols);
#else
  cuda_ccl_copy_buf_send_kernel<<<blocks,threadsPerBlock>>>(a_dev, buf_send_dev, l_rows, l_cols, lld_buf, nblk,
                                                            i_block_loc_fine, j_block_loc_fine, np_fine, np_bc_fine, 
                                                            np_rows_fine, np_cols_fine, np_rows, np_cols);
#endif

  if (debug)
    {
    hipDeviceSynchronize();
    hipError_t cuerr = hipGetLastError();
    if (cuerr != hipSuccess){
      printf("Error in executing cuda_ccl_copy_buf_send: %s\n",hipGetErrorString(cuerr));
    }
  }
}

extern "C" void cuda_ccl_copy_buf_send_FromC(char dataType, intptr_t a_dev, intptr_t buf_send_dev, 
                                             int *l_rows_in, int *l_cols_in, int *lld_buf_in, int *nblk_in,
                                             int *i_block_loc_fine_in, int *j_block_loc_fine_in, int *np_fine_in, int *np_bc_fine_in, 
                                             int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *SM_count_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_ccl_copy_buf_send<double>((double *) a_dev, (double *) buf_send_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_in, j_block_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='S') cuda_ccl_copy_buf_send<float> ((float  *) a_dev, (float  *) buf_send_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_in, j_block_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='Z') cuda_ccl_copy_buf_send<hipDoubleComplex>((hipDoubleComplex *) a_dev, (hipDoubleComplex *) buf_send_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_in, j_block_loc_fine_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='C') cuda_ccl_copy_buf_send<hipFloatComplex> ((hipFloatComplex  *) a_dev, (hipFloatComplex  *) buf_send_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_in, j_block_loc_fine_in, np_fine_in, np_bc_fine_in,
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
}

//________________________________________________________________

template <typename T>
__global__ void cuda_ccl_copy_buf_recv_kernel(T *at_col_dev, T *buf_recv_dev, int l_rows, int l_cols, int lld_buf, int nblk,
                                              int i_block_loc_fine_max, int j_block_loc_fine_max, int np_fine, int np_bc_fine, 
                                              int np_rows_fine, int np_cols_fine, int np_rows, int np_cols) {

  // do i_block_loc_fine = 0, i_block_loc_fine_max
  //   i_block_loc = (np + i_block_loc_fine*np_rows_fine)/np_rows
  //   nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk)

  //   do j_block_loc_fine = 0, j_block_loc_fine_max
  //     j_block_loc = (np_t + j_block_loc_fine*np_cols_fine)/np_cols
  //     nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk)
      
  //     at(1+ i_block_loc     *nblk: nblk_cut_row + i_block_loc     *nblk,   &
  //        1+ j_block_loc     *nblk: nblk_cut_col + j_block_loc     *nblk) = &
  //     transpose(buf_recv(1+ j_block_loc_fine*nblk: nblk_cut_col + j_block_loc_fine*nblk,   &
  //                        1+ i_block_loc_fine*nblk: nblk_cut_row + i_block_loc_fine*nblk))
  //   enddo ! j_block_loc_fine
  // enddo ! i_block_loc_fine

  int di0 = threadIdx.x; // di = 0..nblk_cut_row-1
  int dj0 = blockIdx.x ; // dj = 0..nblk_cut_col-1

  int i_block_loc, j_block_loc, nblk_cut_row, nblk_cut_col;

  int i_block_loc_fine = 0;
  for (; i_block_loc_fine <= i_block_loc_fine_max; i_block_loc_fine++) 
    {
    i_block_loc = (np_fine + i_block_loc_fine*np_rows_fine)/np_rows;
    nblk_cut_row = min(nblk, l_rows-i_block_loc*nblk);

    int j_block_loc_fine = 0;
    for (; j_block_loc_fine <= j_block_loc_fine_max; j_block_loc_fine++) 
      {
      j_block_loc = (np_bc_fine + j_block_loc_fine*np_cols_fine)/np_cols;
      nblk_cut_col = min(nblk, l_cols-j_block_loc*nblk);
      
      for (int dj = dj0; dj < nblk_cut_col; dj += gridDim.x)
        {
        for (int di = di0; di < nblk_cut_row; di += blockDim.x)
          {
          at_col_dev[(di+i_block_loc*     nblk) + (dj+j_block_loc*     nblk)*l_rows] 
      = buf_recv_dev[(dj+j_block_loc_fine*nblk) + (di+i_block_loc_fine*nblk)*lld_buf];
          }
        }
      }
    }

}

template <typename T>
void cuda_ccl_copy_buf_recv(T *at_col_dev, T *buf_recv_dev, int *l_rows_in, int *l_cols_in, int *lld_buf_in, int *nblk_in,
                            int *i_block_loc_fine_max_in, int *j_block_loc_fine_max_in, int *np_fine_in, int *np_bc_fine_in, 
                            int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *SM_count_in, int *debug_in, hipStream_t my_stream){

  int l_rows = *l_rows_in;
  int l_cols = *l_cols_in;
  int lld_buf = *lld_buf_in;
  int nblk = *nblk_in;
  int i_block_loc_fine_max = *i_block_loc_fine_max_in;
  int j_block_loc_fine_max = *j_block_loc_fine_max_in;
  int np_fine = *np_fine_in;
  int np_bc_fine = *np_bc_fine_in;
  int np_rows_fine = *np_rows_fine_in;
  int np_cols_fine = *np_cols_fine_in;
  int np_rows = *np_rows_in;
  int np_cols = *np_cols_in;
  int SM_count = *SM_count_in;
  int debug = *debug_in;

  dim3 blocks = dim3(SM_count, 1, 1);
  dim3 threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK, 1, 1);

#ifdef WITH_GPU_STREAMS
  cuda_ccl_copy_buf_recv_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(at_col_dev, buf_recv_dev, l_rows, l_cols, lld_buf, nblk,
                                                                       i_block_loc_fine_max, j_block_loc_fine_max, np_fine, np_bc_fine, 
                                                                       np_rows_fine, np_cols_fine, np_rows, np_cols);
#else
  cuda_ccl_copy_buf_recv_kernel<<<blocks,threadsPerBlock>>>(at_col_dev, buf_recv_dev, l_rows, l_cols, lld_buf, nblk,
                                                           i_block_loc_fine_max, j_block_loc_fine_max, np_fine, np_bc_fine, 
                                                           np_rows_fine, np_cols_fine, np_rows, np_cols);
#endif
  
    if (debug)
      {
      hipDeviceSynchronize();
      hipError_t cuerr = hipGetLastError();
      if (cuerr != hipSuccess){
        printf("Error in executing cuda_ccl_copy_buf_recv: %s\n",hipGetErrorString(cuerr));
      }
    }
  }

extern "C" void cuda_ccl_copy_buf_recv_FromC(char dataType, intptr_t at_col_dev, intptr_t buf_recv_dev, 
                                             int *l_rows_in, int *l_cols_in, int *lld_buf_in, int *nblk_in,
                                             int *i_block_loc_fine_max_in, int *j_block_loc_fine_max_in, int *np_fine_in, int *np_bc_fine_in, 
                                             int *np_rows_fine_in, int *np_cols_fine_in, int *np_rows_in, int *np_cols_in, int *SM_count_in, int *debug_in, hipStream_t my_stream){
  if (dataType=='D') cuda_ccl_copy_buf_recv<double>((double *) at_col_dev, (double *) buf_recv_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_max_in, j_block_loc_fine_max_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='S') cuda_ccl_copy_buf_recv<float> ((float  *) at_col_dev, (float  *) buf_recv_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_max_in, j_block_loc_fine_max_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='Z') cuda_ccl_copy_buf_recv<hipDoubleComplex>((hipDoubleComplex *) at_col_dev, (hipDoubleComplex *) buf_recv_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_max_in, j_block_loc_fine_max_in, np_fine_in, np_bc_fine_in, 
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
  if (dataType=='C') cuda_ccl_copy_buf_recv<hipFloatComplex> ((hipFloatComplex  *) at_col_dev, (hipFloatComplex  *) buf_recv_dev, l_rows_in, l_cols_in, lld_buf_in, nblk_in,
                                                    i_block_loc_fine_max_in, j_block_loc_fine_max_in, np_fine_in, np_bc_fine_in,
                                                    np_rows_fine_in, np_cols_fine_in, np_rows_in, np_cols_in, SM_count_in, debug_in, my_stream);
}

//_________________________________________________________________________________________________
// non-square grid, TN, NT codepath

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_ab_full_tn_kernel(T *a_dev, T *b_dev, T *aux_a_full_dev, T *aux_b_full_dev,
                                                              int l_rows, int l_cols, int nblk_mult_max, int nblk_mult, int nblk,
                                                              int np_ab_fine, int np_rows, int my_prow,
                                                              int np_t_fine , int np_cols, int my_pcol,
                                                              int np_dirs_fine){
  // if (mod(np_t_fine,np_cols) == my_pcol) then
  //   do j_block_loc_fine = 0, nblk_mult_max/nblk-1
  //     j_block_loc = (np_t_fine + j_block_loc_fine*np_dirs_fine)/np_cols
      
  //     do i_block_loc_fine = 0, nblk_mult/nblk-1
  //       i_block_loc = (np_ab_fine + i_block_loc_fine*np_dirs_fine)/np_rows
        
  //       nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk)
  //       nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk)

  //       if (nblk_rows_cut>0 .and. nblk_cols_cut>0) then
  //         aux_a_full(1+i_block_loc_fine*nblk : nblk_rows_cut+i_block_loc_fine*nblk, &
  //                    1+j_block_loc_fine*nblk : nblk_cols_cut+j_block_loc_fine*nblk) = &
  //                 a (1+i_block_loc*nblk      : nblk_rows_cut+i_block_loc*nblk, &
  //                    1+j_block_loc*nblk      : nblk_cols_cut+j_block_loc*nblk)
  //       endif

  //     enddo ! i_block_loc_fine
  //   enddo ! j_block_loc_fine
  // endif ! (mod(np_t_fine,np_cols) == my_pcol)

  int di0 = threadIdx.x; // 0..nblk
  int dj0 = blockIdx.x ; // 0..nblk

  T Zero = elpaDeviceNumber<T>(0.0);

  int i_block_loc, j_block_loc, i_block_loc_fine, j_block_loc_fine,  nblk_rows_cut, nblk_cols_cut, di, dj;

  if (np_t_fine%np_cols == my_pcol) 
    {
    for (j_block_loc_fine=0; j_block_loc_fine<nblk_mult_max/nblk; j_block_loc_fine++) 
      {
      j_block_loc = (np_t_fine + j_block_loc_fine*np_dirs_fine)/np_cols;
      
      for (i_block_loc_fine=0; i_block_loc_fine<nblk_mult/nblk; i_block_loc_fine++) 
        {
        i_block_loc = (np_ab_fine + i_block_loc_fine*np_dirs_fine)/np_rows;
        
        nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk);
        nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk);

        if (nblk_rows_cut>0 && nblk_cols_cut>0) 
          {
          for (dj=dj0; dj<nblk_cols_cut; dj += gridDim.x)
            for (di=di0; di<nblk_rows_cut; di += blockDim.x)
              aux_a_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk)*nblk_mult] 
                     = a_dev[di + i_block_loc*nblk + (dj + j_block_loc*nblk)*l_rows];
          }
        }
      }
    }

  // do dnp_ab_t = 0, np_dirs_fine/np_cols-1
  //   np_ab_t_fine = dnp_ab_t*np_cols + my_pcol

  //   do j_block_loc_fine = 0, nblk_mult_max/nblk-1
  //     j_block_loc = (np_ab_t_fine + j_block_loc_fine*np_dirs_fine)/np_cols
      
  //     do i_block_loc_fine = 0, nblk_mult/nblk-1
  //       i_block_loc = (np_ab_fine + i_block_loc_fine*np_dirs_fine)/np_rows

  //       nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk)
  //       nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk)
        
  //       if (nblk_rows_cut>0 .and. nblk_cols_cut>0) then
  //         aux_b_full(1+i_block_loc_fine*nblk : nblk_rows_cut+i_block_loc_fine*nblk, &
  //                   1            +j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max : &
  //                   nblk_cols_cut+j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max) = &
  //                   b(1+i_block_loc*nblk      :nblk_rows_cut+i_block_loc*nblk, &
  //                     1+j_block_loc*nblk      :nblk_cols_cut+j_block_loc*nblk)
  //       endif

  //       ! nullify the unused part of the block in b
  //       if (nblk_rows_cut<nblk) then
  //         if (nblk_rows_cut>0) then
  //           aux_b_full(nblk_rows_cut+1+i_block_loc_fine*nblk : nblk+i_block_loc_fine*nblk, &
  //                       1            +j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max : &
  //                       nblk_cols_cut+j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max) = 0
  //         else ! for negative nblk_rows_cut we nullify the whole block (it's locally absent)
  //           aux_b_full(1+i_block_loc_fine*nblk : nblk+i_block_loc_fine*nblk, &
  //                       1   +j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max : &
  //                       nblk+j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max) = 0
  //         endif
  //       endif

  //       if (nblk_cols_cut<nblk) then
  //         if (nblk_cols_cut>0) then
  //           aux_b_full(1+i_block_loc_fine*nblk : nblk_rows_cut+i_block_loc_fine*nblk, &
  //                       nblk_cols_cut+1+j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max : &
  //                       nblk           +j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max) = 0
  //         else
  //           aux_b_full(1+i_block_loc_fine*nblk : nblk+i_block_loc_fine*nblk, &
  //                       1   +j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max : &
  //                       nblk+j_block_loc_fine*nblk+dnp_ab_t*nblk_mult_max) = 0
  //         endif
  //       endif

  //     enddo ! i_block_loc_fine
  //   enddo ! j_block_loc_fine
  // enddo ! np_ab_t_fine

  int dnp_ab_t, np_ab_t_fine;
  for (dnp_ab_t = 0; dnp_ab_t < np_dirs_fine/np_cols; dnp_ab_t++)
    {
    np_ab_t_fine = dnp_ab_t*np_cols + my_pcol;
    for (j_block_loc_fine = 0; j_block_loc_fine < nblk_mult_max/nblk; j_block_loc_fine++)
      {
      j_block_loc = (np_ab_t_fine + j_block_loc_fine*np_dirs_fine)/np_cols;

      for (i_block_loc_fine = 0; i_block_loc_fine < nblk_mult/nblk; i_block_loc_fine++)
        {
        i_block_loc = (np_ab_fine + i_block_loc_fine*np_dirs_fine)/np_rows;

        nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk);
        nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk);

        if (nblk_rows_cut>0 && nblk_cols_cut>0)
          {
          for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
            for (di = di0; di < nblk_rows_cut; di += blockDim.x)
              aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max)*nblk_mult] 
                    = b_dev[di + i_block_loc*nblk + (dj + j_block_loc*nblk)*l_rows];
          }

        // nullify the unused part of the block in b
        if (nblk_rows_cut < nblk)
          {
          if (nblk_rows_cut > 0)
            {
            for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
              for (di = nblk_rows_cut+di0; di < nblk; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max)*nblk_mult] = Zero;
            }
          else // for negative nblk_rows_cut we nullify the whole block (it's locally absent)
            {
            for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
              for (di = di0; di < nblk; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max)*nblk_mult] = Zero;
            }
          }

        if (nblk_cols_cut < nblk)
          {
          if (nblk_cols_cut > 0)
            {
            for (dj = nblk_cols_cut+dj0; dj < nblk; dj += gridDim.x)
              for (di = di0; di < nblk_rows_cut; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max)*nblk_mult] = Zero;
            }
          else 
            {
            for (dj = dj0; dj < nblk; dj += gridDim.x)
              for (di = di0; di < nblk_rows_cut; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max)*nblk_mult] = Zero;
            }
          }
       }
        
      }
    }
}

template <typename T>
__global__ void cuda_copy_and_set_zeros_aux_ab_full_nt_kernel(T *a_dev, T *b_dev, T *aux_a_full_dev, T *aux_b_full_dev,
                                                              int l_rows, int l_cols, int nblk_mult_max, int nblk_mult, int nblk,
                                                              int np_ab_fine, int np_rows, int my_prow,
                                                              int np_t_fine , int np_cols, int my_pcol,
                                                              int np_dirs_fine){
  int di0 = threadIdx.x;
  int dj0 = blockIdx.x;

  T Zero = elpaDeviceNumber<T>(0.0);

  int i_block_loc, j_block_loc, i_block_loc_fine, j_block_loc_fine, nblk_rows_cut, nblk_cols_cut, di, dj;
  int dnp_ab_t, np_ab_t_fine;

  if (np_t_fine%np_rows == my_prow) 
    {
    for (i_block_loc_fine=0; i_block_loc_fine<nblk_mult_max/nblk; i_block_loc_fine++) 
      {
      i_block_loc = (np_t_fine + i_block_loc_fine*np_dirs_fine)/np_rows;

      for (j_block_loc_fine=0; j_block_loc_fine<nblk_mult/nblk; j_block_loc_fine++) 
        {
        j_block_loc = (np_ab_fine + j_block_loc_fine*np_dirs_fine)/np_cols;

        nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk);
        nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk);

        if (nblk_rows_cut> 0 && nblk_cols_cut> 0)
          {
          for (dj=dj0; dj<nblk_cols_cut; dj += gridDim.x)
            for (di=di0; di<nblk_rows_cut; di += blockDim.x)          
              aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk)*nblk_mult] =
                       b_dev[di + i_block_loc*nblk + (dj + j_block_loc*nblk)*l_rows];
          }

        // Nullify the unused part of the block in b
        if (nblk_rows_cut < nblk)
          {
          if (nblk_rows_cut > 0)
            {
            for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
              for (di = nblk_rows_cut + di0; di < nblk; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk)*nblk_mult] = Zero;
            }
          else // For negative nblk_rows_cut we nullify the whole block  (it's locally absent)
            {
            for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
              for (di = di0; di < nblk; di += blockDim.x)            
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine*nblk)*nblk_mult] = Zero;
            }
          }
        

        if (nblk_cols_cut < nblk)
          {
          if (nblk_cols_cut > 0)
            {
            for (dj = nblk_cols_cut + dj0; dj < nblk; dj += gridDim.x)
              for (di = di0; di < nblk_rows_cut; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine * nblk)*nblk_mult] = Zero;
            }
          else
            {
            for (dj = dj0; dj < nblk; dj += gridDim.x)
              for (di = di0; di < nblk_rows_cut; di += blockDim.x)
                aux_b_full_dev[di + i_block_loc_fine*nblk + (dj + j_block_loc_fine * nblk)*nblk_mult] = Zero;
            }
          }
        }
      }
    }


  for (dnp_ab_t = 0; dnp_ab_t < np_dirs_fine/np_rows; dnp_ab_t++)
    {
    np_ab_t_fine = dnp_ab_t*np_rows + my_prow;

    for (i_block_loc_fine = 0; i_block_loc_fine < nblk_mult_max/nblk; i_block_loc_fine++)
      {
      i_block_loc = (np_ab_t_fine + i_block_loc_fine*np_dirs_fine)/np_rows;

      for (j_block_loc_fine = 0; j_block_loc_fine < nblk_mult/nblk; j_block_loc_fine++)
        {
        j_block_loc = (np_ab_fine + j_block_loc_fine*np_dirs_fine)/np_cols;

        nblk_rows_cut = min(nblk, l_rows - i_block_loc*nblk);
        nblk_cols_cut = min(nblk, l_cols - j_block_loc*nblk);

        if (nblk_rows_cut > 0 && nblk_cols_cut > 0)
          {
          for (dj = dj0; dj < nblk_cols_cut; dj += gridDim.x)
            for (di = di0; di < nblk_rows_cut; di += blockDim.x)
              aux_a_full_dev[di + i_block_loc_fine*nblk + dnp_ab_t*nblk_mult_max + (dj + j_block_loc_fine*nblk)*nblk_mult] =
                       a_dev[di + i_block_loc*nblk + (dj + j_block_loc*nblk)*l_rows];
                    
                
          }
        }
      }
    }
}

template <typename T>
void cuda_copy_and_set_zeros_aux_ab_full_tn_nt(int *a_transoposed_in, T *a_dev, T *b_dev, T *aux_a_full_dev, T *aux_b_full_dev,
                                            int *l_rows_in, int *l_cols_in, int *nblk_mult_max_in, int *nblk_mult_in, int *nblk_in,
                                            int *np_ab_fine_in, int *np_rows_in, int *my_prow_in,
                                            int *np_t_fine_in, int *np_cols_in, int *my_pcol_in,
                                            int *np_dirs_fine_in,int *SM_count_in,
                                            int *debug_in, hipStream_t my_stream){
    
    int a_transoposed = *a_transoposed_in;
    int l_rows = *l_rows_in;
    int l_cols = *l_cols_in;
    int nblk_mult_max = *nblk_mult_max_in;
    int nblk_mult = *nblk_mult_in;
    int nblk = *nblk_in;
    int np_ab_fine = *np_ab_fine_in;
    int np_rows = *np_rows_in;
    int my_prow = *my_prow_in;
    int np_t_fine = *np_t_fine_in;
    int np_cols = *np_cols_in;
    int my_pcol = *my_pcol_in;
    int np_dirs_fine = *np_dirs_fine_in;
    int SM_count = *SM_count_in;
    int debug = *debug_in;

    dim3 blocksPerGrid(SM_count, 1, 1); 
    dim3 threadsPerBlock(min(nblk, MAX_THREADS_PER_BLOCK/2), 1, 1); // use only half of the max threads due to high register usage

    if (a_transoposed)
      {
#ifdef WITH_GPU_STREAMS
      cuda_copy_and_set_zeros_aux_ab_full_tn_kernel<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(
          a_dev, b_dev, aux_a_full_dev, aux_b_full_dev,
          l_rows, l_cols, nblk_mult_max, nblk_mult, nblk,
          np_ab_fine, np_rows, my_prow,
          np_t_fine, np_cols, my_pcol,
          np_dirs_fine);
#else
      cuda_copy_and_set_zeros_aux_ab_full_tn_kernel<<<blocksPerGrid, threadsPerBlock>>>(
          a_dev, b_dev, aux_a_full_dev, aux_b_full_dev,
          l_rows, l_cols, nblk_mult_max, nblk_mult, nblk,
          np_ab_fine, np_rows, my_prow,
          np_t_fine, np_cols, my_pcol,
          np_dirs_fine);
#endif
      }
    else 
      {
#ifdef WITH_GPU_STREAMS
      cuda_copy_and_set_zeros_aux_ab_full_nt_kernel<<<blocksPerGrid, threadsPerBlock, 0, my_stream>>>(
          a_dev, b_dev, aux_a_full_dev, aux_b_full_dev,
          l_rows, l_cols, nblk_mult_max, nblk_mult, nblk,
          np_ab_fine, np_rows, my_prow,
          np_t_fine, np_cols, my_pcol,
          np_dirs_fine);
#else
      cuda_copy_and_set_zeros_aux_ab_full_nt_kernel<<<blocksPerGrid, threadsPerBlock>>>(
          a_dev, b_dev, aux_a_full_dev, aux_b_full_dev,
          l_rows, l_cols, nblk_mult_max, nblk_mult, nblk,
          np_ab_fine, np_rows, my_prow,
          np_t_fine, np_cols, my_pcol,
          np_dirs_fine);
#endif
      }
    if (debug)
    {
        hipDeviceSynchronize();
        hipError_t cuerr = hipGetLastError();
        if (cuerr != hipSuccess)
        {
            printf("Error in executing cuda_copy_and_set_zeros_aux_ab_full_tn: %s\n", hipGetErrorString(cuerr));
        }
    }
}

extern "C" void cuda_copy_and_set_zeros_aux_ab_full_tn_nt_FromC(char dataType, int *a_transoposed_in, intptr_t a_dev, intptr_t b_dev, intptr_t aux_a_full_dev, intptr_t aux_b_full_dev,
                                                             int *l_rows_in, int *l_cols_in, int *nblk_mult_max_in, int *nblk_mult_in, int *nblk_in,
                                                             int *np_ab_fine_in, int *np_rows_in, int *my_prow_in,
                                                             int *np_t_fine_in , int *np_cols_in, int *my_pcol_in,
                                                             int *np_dirs_fine_in,
                                                             int *SM_count_in, int *debug_in, hipStream_t my_stream){
  if (dataType == 'D') cuda_copy_and_set_zeros_aux_ab_full_tn_nt<double>(a_transoposed_in, (double *)a_dev, (double *)b_dev, (double *)aux_a_full_dev, (double *)aux_b_full_dev,
                                                       l_rows_in, l_cols_in, nblk_mult_max_in, nblk_mult_in, nblk_in,
                                                       np_ab_fine_in, np_rows_in, my_prow_in,
                                                       np_t_fine_in , np_cols_in, my_pcol_in,
                                                       np_dirs_fine_in,
                                                       SM_count_in, debug_in, my_stream);
  if (dataType == 'S') cuda_copy_and_set_zeros_aux_ab_full_tn_nt<float>(a_transoposed_in, (float *)a_dev, (float *)b_dev, (float *)aux_a_full_dev, (float *)aux_b_full_dev,
                                                      l_rows_in, l_cols_in, nblk_mult_max_in, nblk_mult_in, nblk_in,
                                                      np_ab_fine_in, np_rows_in, my_prow_in,
                                                      np_t_fine_in , np_cols_in, my_pcol_in,
                                                      np_dirs_fine_in,
                                                      SM_count_in, debug_in, my_stream);
  if (dataType == 'Z') cuda_copy_and_set_zeros_aux_ab_full_tn_nt<hipDoubleComplex>(a_transoposed_in, (hipDoubleComplex *)a_dev, (hipDoubleComplex *)b_dev, (hipDoubleComplex *)aux_a_full_dev, (hipDoubleComplex *)aux_b_full_dev,
                                                                l_rows_in, l_cols_in, nblk_mult_max_in, nblk_mult_in, nblk_in,
                                                                np_ab_fine_in, np_rows_in, my_prow_in,
                                                                np_t_fine_in , np_cols_in, my_pcol_in,
                                                                np_dirs_fine_in,
                                                                SM_count_in, debug_in, my_stream);
  if (dataType == 'C') cuda_copy_and_set_zeros_aux_ab_full_tn_nt<hipFloatComplex>(a_transoposed_in, (hipFloatComplex *)a_dev, (hipFloatComplex *)b_dev, (hipFloatComplex *)aux_a_full_dev, (hipFloatComplex *)aux_b_full_dev,
                                                               l_rows_in, l_cols_in, nblk_mult_max_in, nblk_mult_in, nblk_in,
                                                               np_ab_fine_in, np_rows_in, my_prow_in,
                                                               np_t_fine_in , np_cols_in, my_pcol_in,
                                                               np_dirs_fine_in,
                                                               SM_count_in, debug_in, my_stream);
}

//________________________________________________________________


