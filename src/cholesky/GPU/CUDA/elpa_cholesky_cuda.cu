#include "hip/hip_runtime.h"
//    Copyright 2021, A. Marek
//
//    This file is part of ELPA.
//
//    The ELPA library was originally created by the ELPA consortium,
//    consisting of the following organizations:
//
//    - Max Planck Computing and Data Facility (MPCDF), formerly known as
//      Rechenzentrum Garching der Max-Planck-Gesellschaft (RZG),
//    - Bergische Universität Wuppertal, Lehrstuhl für angewandte
//      Informatik,
//    - Technische Universität München, Lehrstuhl für Informatik mit
//      Schwerpunkt Wissenschaftliches Rechnen ,
//    - Fritz-Haber-Institut, Berlin, Abt. Theorie,
//    - Max-Plack-Institut für Mathematik in den Naturwissenschaften,
//      Leipzig, Abt. Komplexe Strukutren in Biologie und Kognition,
//      and
//    - IBM Deutschland GmbH
//
//    This particular source code file contains additions, changes and
//    enhancements authored by Intel Corporation which is not part of
//    the ELPA consortium.
//
//    More information can be found here:
//    http://elpa.mpcdf.mpg.de/
//
//    ELPA is free software: you can redistribute it and/or modify
//    it under the terms of the version 3 of the license of the
//    GNU Lesser General Public License as published by the Free
//    Software Foundation.
//
//    ELPA is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU Lesser General Public License for more details.
//
//    You should have received a copy of the GNU Lesser General Public License
//    along with ELPA.  If not, see <http://www.gnu.org/licenses/>
//
//    ELPA reflects a substantial effort on the part of the original
//    ELPA consortium, and we ask you to respect the spirit of the
//    license that we chose: i.e., please contribute any changes you
//    may have back to the original ELPA library distribution, and keep
//    any derivatives of ELPA under the same license that we chose for
//    the original distribution, the GNU Lesser General Public License.
//
//    This file was written by A. Marek, MPCDF

#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <alloca.h>
#include <complex.h>
#include <hip/hip_complex.h>
#include <stdint.h>
#include <assert.h>
#include "config-f90.h"

#define errormessage(x, ...) do { fprintf(stderr, "%s:%d " x, __FILE__, __LINE__, __VA_ARGS__ ); } while (0)

__global__ void cuda_check_device_info_kernel(int *info_dev){
  // if (*info_dev != 0){
  //   printf("Error in executing check_device_info_kerne: %d\n", *info_dev);
  // }
  assert(*info_dev == 0);
}

extern "C" void cuda_check_device_info_FromC(int *info_dev, hipStream_t my_stream){

  dim3 blocks = dim3(1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_check_device_info_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(info_dev);
#else
  cuda_check_device_info_kernel<<<blocks,threadsPerBlock>>>(info_dev);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing check_device_info_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_accumulate_device_info_kernel(int *info_abs_dev, int *info_new_dev){
  *info_abs_dev += abs(*info_new_dev);
}

extern "C" void cuda_accumulate_device_info_FromC(int *info_abs_dev, int *info_new_dev, hipStream_t my_stream){

  dim3 blocks = dim3(1,1,1);
  dim3 threadsPerBlock = dim3(1,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_accumulate_device_info_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(info_abs_dev, info_new_dev);
#else
  cuda_accumulate_device_info_kernel<<<blocks,threadsPerBlock>>>(info_abs_dev, info_new_dev);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing accumulate_device_info_kernel: %s\n",hipGetErrorString(cuerr));
  }
}

__global__ void cuda_copy_double_a_tmatc_kernel(double *a_dev, double *tmatc_dev, const int l_cols, const int matrixRows, const int l_colx, const int l_row1, const int nblk){

  int ii_index    = threadIdx.x +1; // range 1..nblk
  int jj_index = blockIdx.x + 1; // range 1..l_cols-l_colx+1
  tmatc_dev[l_colx-1+jj_index-1+(ii_index-1)*l_cols] = a_dev[l_row1-1+ii_index-1 + (l_colx-1+jj_index-1)*matrixRows];
}

extern "C" void cuda_copy_double_a_tmatc_FromC(double *a_dev, double *tmatc_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, hipStream_t my_stream){
  int nblk = *nblk_in;   
  int matrixRows = *matrixRows_in;
  int l_cols = *l_cols_in;
  int l_colx = *l_colx_in;
  int l_row1 = *l_row1_in;

//#ifdef WITH_GPU_STREAMS
//  hipStream_t streamId = *((hipStream_t*)my_stream);
//#endif

  dim3 blocks = dim3(l_cols-l_colx+1,1,1);
  dim3 threadsPerBlock = dim3(nblk,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_a_tmatc_kernel<<<blocks,threadsPerBlock, 0, my_stream>>>(a_dev, tmatc_dev, l_cols, matrixRows, l_colx, l_row1, nblk);
#else
  cuda_copy_double_a_tmatc_kernel<<<blocks,threadsPerBlock>>>(a_dev, tmatc_dev, l_cols, matrixRows, l_colx, l_row1, nblk);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_a_tmatc_kernel: %s\n",hipGetErrorString(cuerr));
    printf("blocks=%d, threadsPerBlock=%d \n", l_cols-l_colx+1, nblk);
  }
}

__global__ void cuda_copy_float_a_tmatc_kernel(float *a_dev, float *tmatc_dev, const int l_cols, const int matrixRows, const int l_colx, const int l_row1, const int nblk){

  int ii_index    = threadIdx.x +1; // range 1..nblk
  int jj_index = blockIdx.x + 1; // range 1..l_cols-l_colx+1
  tmatc_dev[l_colx-1+jj_index-1+(ii_index-1)*l_cols] = a_dev[l_row1-1+ii_index-1 + (l_colx-1+jj_index-1)*matrixRows];
}

extern "C" void cuda_copy_float_a_tmatc_FromC(float *a_dev, float *tmatc_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, hipStream_t my_stream){
  int nblk = *nblk_in;   
  int matrixRows = *matrixRows_in;
  int l_cols = *l_cols_in;
  int l_colx = *l_colx_in;
  int l_row1 = *l_row1_in;

//#ifdef WITH_GPU_STREAMS
//  hipStream_t streamId = *((hipStream_t*)my_stream);
//#endif

  dim3 blocks = dim3(l_cols-l_colx+1,1,1);
  dim3 threadsPerBlock = dim3(nblk,1,1);

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_a_tmatc_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_dev, tmatc_dev, l_cols, matrixRows, l_colx, l_row1, nblk);
#else
  cuda_copy_float_a_tmatc_kernel<<<blocks,threadsPerBlock>>>(a_dev, tmatc_dev, l_cols, matrixRows, l_colx, l_row1, nblk);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_a_tmatc_kernel: %s\n",hipGetErrorString(cuerr));
    printf("blocks=%d, threadsPerBlock=%d \n", l_cols-l_colx+1, nblk);
  }
}

__global__ void cuda_copy_double_complex_a_tmatc_kernel(hipDoubleComplex *a_dev, hipDoubleComplex *tmatc_dev, const int l_cols, const int matrixRows, const int l_colx, const int l_row1){

  int ii_index    = threadIdx.x +1; // range 1..nblk
  int jj_index = blockIdx.x + 1; // range 1..l_cols-l_colx+1
  tmatc_dev[l_colx-1+jj_index-1+(ii_index-1)*l_cols] = hipConj(a_dev[l_row1-1+ii_index-1 + (l_colx-1+jj_index-1)*matrixRows]);
}

extern "C" void cuda_copy_double_complex_a_tmatc_FromC(double _Complex *a_dev, double _Complex *tmatc_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, hipStream_t my_stream){
  int nblk = *nblk_in;   
  int matrixRows = *matrixRows_in;
  int l_cols = *l_cols_in;
  int l_colx = *l_colx_in;
  int l_row1 = *l_row1_in;

//#ifdef WITH_GPU_STREAMS
//  hipStream_t streamId = *((hipStream_t*)my_stream);
//#endif

  dim3 blocks = dim3(l_cols-l_colx+1,1,1);
  dim3 threadsPerBlock = dim3(nblk,1,1);

  hipDoubleComplex* a_casted = (hipDoubleComplex*) a_dev;
  hipDoubleComplex* tmatc_casted = (hipDoubleComplex*) tmatc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_double_complex_a_tmatc_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_casted, tmatc_casted, l_cols, matrixRows, l_colx, l_row1);
#else
  cuda_copy_double_complex_a_tmatc_kernel<<<blocks,threadsPerBlock>>>(a_casted, tmatc_casted, l_cols, matrixRows, l_colx, l_row1);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_double_complex_a_tmatc_kernel: %s\n",hipGetErrorString(cuerr));
    printf("blocks=%d, threadsPerBlock=%d \n", l_cols-l_colx+1, nblk);
  }
}

__global__ void cuda_copy_float_complex_a_tmatc_kernel(hipFloatComplex *a_dev, hipFloatComplex *tmatc_dev, const int l_cols, const int matrixRows, const int l_colx, const int l_row1){

  int ii_index    = threadIdx.x +1; // range 1..nblk
  int jj_index = blockIdx.x + 1; // range 1..l_cols-l_colx+1
  tmatc_dev[l_colx-1+jj_index-1+(ii_index-1)*l_cols] = hipConjf(a_dev[l_row1-1+ii_index-1 + (l_colx-1+jj_index-1)*matrixRows]);
}

extern "C" void cuda_copy_float_complex_a_tmatc_FromC(float _Complex *a_dev, float _Complex *tmatc_dev, int *nblk_in, int *matrixRows_in, int *l_cols_in, int *l_colx_in, int *l_row1_in, hipStream_t my_stream){
  int nblk = *nblk_in;   
  int matrixRows = *matrixRows_in;
  int l_cols = *l_cols_in;
  int l_colx = *l_colx_in;
  int l_row1 = *l_row1_in;

//#ifdef WITH_GPU_STREAMS
//  hipStream_t streamId = *((hipStream_t*)my_stream);
//#endif

  dim3 blocks = dim3(l_cols-l_colx+1,1,1);
  dim3 threadsPerBlock = dim3(nblk,1,1);

  hipFloatComplex* a_casted = (hipFloatComplex*) a_dev;
  hipFloatComplex* tmatc_casted = (hipFloatComplex*) tmatc_dev;

#ifdef WITH_GPU_STREAMS
  cuda_copy_float_complex_a_tmatc_kernel<<<blocks,threadsPerBlock,0,my_stream>>>(a_casted, tmatc_casted, l_cols, matrixRows, l_colx, l_row1);
#else
  cuda_copy_float_complex_a_tmatc_kernel<<<blocks,threadsPerBlock>>>(a_casted, tmatc_casted, l_cols, matrixRows, l_colx, l_row1);
#endif
  hipError_t cuerr = hipGetLastError();
  if (cuerr != hipSuccess){
    printf("Error in executing copy_float_complex_a_tmatc_kernel: %s\n",hipGetErrorString(cuerr));
    printf("blocks=%d, threadsPerBlock=%d \n", l_cols-l_colx+1, nblk); 
  }
}
